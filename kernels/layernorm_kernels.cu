#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>

#include "dispatch_utils.h"
#include "quant_utils.cuh"
#include "reduction_utils.cuh"

namespace aphrodite {

// TODO: Further optimize this kernel.
template <typename scalar_t>
__global__ void
rms_norm_kernel(scalar_t *__restrict__ out,         // [num_tokens, hidden_size]
                const scalar_t *__restrict__ input, // [num_tokens, hidden_size]
                const scalar_t *__restrict__ weight, // [hidden_size]
                const float epsilon, const int num_tokens,
                const int hidden_size) {
  __shared__ float s_variance;
  float variance = 0.0f;

  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    const float x = (float)input[blockIdx.x * hidden_size + idx];
    variance += x * x;
  }
  variance = blockReduceSum<float>(variance);
  if (threadIdx.x == 0) {
    s_variance = rsqrtf(variance / hidden_size + epsilon);
  }
  __syncthreads();

  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float x = (float)input[blockIdx.x * hidden_size + idx];
    out[blockIdx.x * hidden_size + idx] =
        ((scalar_t)(x * s_variance)) * weight[idx];
  }
}

template <typename T>
__global__ void RMSLayerNorm(const T *__restrict input,
                             const T *__restrict gamma, int8_t *output,
                             const float layernorm_eps, int m, int n) {
  // layernorm module in the T5 style No bias and no subtraction of mean.
  const int tid = threadIdx.x;

  __shared__ float s_variance;
  float variance = 0.0f;

  float local_var_sum = 0.0f;
  for (int i = tid; i < n; i += blockDim.x) {
    // float diff = (float)(ldg(&input[blockIdx.x * n + i]));
    float diff = (float)(input[blockIdx.x * n + i]);
    local_var_sum += diff * diff;
  }
  variance = blockReduceSum(local_var_sum);

  if (threadIdx.x == 0) {
    s_variance = rsqrtf(variance / (float)n + layernorm_eps);
  }
  __syncthreads();

  for (int i = tid; i < n; i += blockDim.x) {
    output[blockIdx.x * n + i] =
        // float_to_int8_rn((((float)input[blockIdx.x * n + i]) * s_variance) *
        //                  (float)(ldg(&gamma[i])));
        float_to_int8_rn((((float)input[blockIdx.x * n + i]) * s_variance) *
                         (float)(gamma[i]));
  }
}

template <typename T>
void invokeRMSLayerNorm(int8_t *out, const T *input, const T *gamma,
                        //   const T*     beta,
                        const float layernorm_eps, const int m, const int n,
                        hipStream_t stream) {
  // if (beta != nullptr) {
  //     invokeGeneralLayerNorm(out, input, gamma, beta, layernorm_eps, m, n,
  //     (float*)nullptr, 0, stream); return;
  // }

  dim3 grid(m);
  dim3 block(min(n, 1024));

  /* For general cases, n is equal to hidden_units, e.g., 512/1024.
      Since we have warp shuffle inside the code, block.x % 32 should be 0.
  */
  if (n % 32 != 0) {
    block.x = 1024;
  }

  block.x =
      block.x / (4 / sizeof(T)); // if using half, only need half of block.x

  /* should pay attention to the rsqrt precision*/
  RMSLayerNorm<T><<<grid, block, 0, stream>>>(input, gamma, out, layernorm_eps,
                                              m, n); // For gpt-3
}

} // namespace aphrodite

void rms_norm(torch::Tensor &out,    // [num_tokens, hidden_size]
              torch::Tensor &input,  // [num_tokens, hidden_size]
              torch::Tensor &weight, // [hidden_size]
              float epsilon) {
  int num_tokens = input.size(0);
  int hidden_size = input.size(1);

  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 1024));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  APHRODITE_DISPATCH_FLOATING_TYPES(input.scalar_type(), "rms_norm_kernel", [&] {
    aphrodite::rms_norm_kernel<scalar_t><<<grid, block, 0, stream>>>(
        out.data_ptr<scalar_t>(), input.data_ptr<scalar_t>(),
        weight.data_ptr<scalar_t>(), epsilon, num_tokens, hidden_size);
  });
}

void invoke_rms_norm_quant(torch::Tensor &out,   // [num_tokens, hidden_size]
                           torch::Tensor &input, // [num_tokens, hidden_size]
                           torch::Tensor &gamma, // [hidden_size]
                           float epsilon) {
  int m = input.size(0);
  int n = input.size(1);
  dim3 grid(m);
  dim3 block(min(n, 1024));

  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  APHRODITE_DISPATCH_FLOATING_TYPES(input.scalar_type(), "invokeRMSLayerNorm", [&] {
    aphrodite::RMSLayerNorm<scalar_t><<<grid, block, 0, stream>>>(
        input.data_ptr<scalar_t>(), gamma.data_ptr<scalar_t>(), out.data_ptr<int8_t>(),
         epsilon, m, n);
  });
}