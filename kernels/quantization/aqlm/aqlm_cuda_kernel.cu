#include "hip/hip_runtime.h"
/*
 * Modified by Neural Magic
 * Adapted from https://github.com/Vahe1994/AQLM
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *         http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

 #include <hip/hip_runtime.h>
 #include <hip/hip_fp16.h>
 #include <hip/hip_runtime.h>
 #include <c10/cuda/CUDAStream.h>
 
 #include <iostream>
 
 __global__ void Code1x16MatVec(
   const int4* __restrict__ A,
   const int4* __restrict__ B,
         int4* __restrict__ C,
   const int4* __restrict__ codebook,
   const int prob_m,
   const int prob_k,
   const int4 codebook_a_sizes,  // cumulative sizes of A spanning each codebook, at most 3 long.
   const int codebook_stride // as int4.
 ) {
   int a_gl_stride = prob_k / 8 / 8;
   int a_gl_rd = (blockDim.x / 32) * blockIdx.x + (threadIdx.x / 32);
   bool pred = a_gl_rd < prob_m;
 
   if (pred)
   {
     // advance to the correct codebook, this easy because we only multiply one column of the codebook.
     auto codebook_size = &codebook_a_sizes.x;
     while (a_gl_rd >= *codebook_size)
     {
         codebook += codebook_stride;
         ++codebook_size;
     }
   }
 
   int b_gl_rd = 0;
   int c_gl_wr = a_gl_rd;
   a_gl_rd = a_gl_stride * a_gl_rd + threadIdx.x % 32;
   int a_gl_end = a_gl_rd + a_gl_stride - threadIdx.x % 32;
 
   __shared__ int4 sh_b[32 * 9];
   float res = 0;
 
   int iters = (prob_k / 8 + 8 * 32 - 1) / (8 * 32);
   while (iters--) {
     // We pad shared memory to avoid bank conflicts during reads
     __syncthreads();
     for (int i = threadIdx.x; i < 32 * 8; i += blockDim.x) {
       if (b_gl_rd + i < prob_k / 8)
         sh_b[9 * (i / 8) + i % 8] = B[b_gl_rd + i];
     }
     __syncthreads();
     b_gl_rd += 32 * 8;
 
     int b_sh_rd = 9 * (threadIdx.x % 32);
     if (pred && a_gl_rd < a_gl_end) {
       const uint16_t* enc = reinterpret_cast<const uint16_t*>(&A[a_gl_rd]);
       #pragma unroll
       for (int i = 0; i < 8; i++) {
         uint32_t dec[4];
         // We bypass the L1 cache to avoid massive amounts of memory streaming that doesn't
         // actually help us; this brings > 2x speedup.
         asm volatile (
           "ld.cg.global.v4.u32 {%0, %1, %2, %3}, [%4];"
           : "=r"(dec[0]), "=r"(dec[1]), "=r"(dec[2]), "=r"(dec[3])
           : "l"((void*) &codebook[enc[i]])
         );
         half2* a = reinterpret_cast<half2*>(&dec);
         half2* b = reinterpret_cast<half2*>(&sh_b[b_sh_rd]);
         half2 res2 = {};
         #pragma unroll
         for (int j = 0; j < 4; j++)
           res2 = __hfma2(a[j], b[j], res2);
         res += __half2float(res2.x) + __half2float(res2.y);
         b_sh_rd++;
       }
       a_gl_rd += 32;
     }
   }
 
   if (pred) {
     #pragma unroll
     for (int i = 16; i > 0; i /= 2)
       res += __shfl_down_sync(0xffffffff, res, i);
     if (threadIdx.x % 32 == 0)
       reinterpret_cast<__half*>(C)[c_gl_wr] = __float2half(res);
   }
 }
 
 __global__ void Code2x8MatVec(
   const int4* __restrict__ A,
   const int4* __restrict__ B,
         int4* __restrict__ C,
   const int4* __restrict__ codebook,
   int prob_m,
   int prob_k,
   const int4 codebook_a_sizes,  // cumulative sizes of A spanning each codebook, at most 3 long.
   const int codebook_stride // as int4.
 
 ) {
   int a_gl_stride = prob_k / 8 / 8;
   int a_gl_rd = (blockDim.x / 32) * blockIdx.x + (threadIdx.x / 32);
   bool pred = a_gl_rd < prob_m;
 
   if (pred)
   {
     // advance to the correct codebook, this easy because we only multiply one column of the codebook.
     auto codebook_size = &codebook_a_sizes.x;
     while (a_gl_rd >= *codebook_size)
     {
         codebook += codebook_stride;
         ++codebook_size;
     }
   }
 
   int b_gl_rd = 0;
   int c_gl_wr = a_gl_rd;
   a_gl_rd = a_gl_stride * a_gl_rd + threadIdx.x % 32;
   int a_gl_end = a_gl_rd + a_gl_stride - threadIdx.x % 32;
   int lane = threadIdx.x % 8;
 
   extern __shared__ int4 sh[];
   int4* sh_b = sh;
   int4* sh_code = sh_b + 32 * 9;
   int4* sh_code0 = sh_code;
   int4* sh_code1 = sh_code + 256 * 8;
 
   for (int i = threadIdx.x; i < 2 * 256; i += blockDim.x) {
     int4 dec = codebook[i];
     #pragma unroll
     for (int j = 0; j < 8; j++)
       sh_code[8 * i + (j + lane) % 8] = dec;
   }
   __syncthreads();
 
   float res = 0;
 
   int iters = (prob_k / 8 + 8 * 32 - 1) / (8 * 32);
   while (iters--) {
     // We pad shared memory to avoid bank conflicts during reads
     __syncthreads();
     for (int i = threadIdx.x; i < 32 * 8; i += blockDim.x) {
       if (b_gl_rd + i < prob_k / 8)
         sh_b[9 * (i / 8) + i % 8] = B[b_gl_rd + i];
     }
     __syncthreads();
     b_gl_rd += 32 * 8;
 
     int b_sh_rd = 9 * (threadIdx.x % 32);
     if (pred && a_gl_rd < a_gl_end) {
       const uint8_t* enc = reinterpret_cast<const uint8_t*>(&A[a_gl_rd]);
       #pragma unroll
       for (int i = 0; i < 8; i++) {
         half2* a0 = reinterpret_cast<half2*>(&sh_code0[8 * enc[2 * i + 0] + lane]);
         half2* a1 = reinterpret_cast<half2*>(&sh_code1[8 * enc[2 * i + 1] + lane]);
         half2*  b = reinterpret_cast<half2*>(&sh_b[b_sh_rd]);
         half2 res2 = {};
         #pragma unroll
         for (int j = 0; j < 4; j++)
           res2 = __hfma2(__hadd2(a0[j], a1[j]), b[j], res2);
         res += __half2float(res2.x) + __half2float(res2.y);
         b_sh_rd++;
       }
       a_gl_rd += 32;
     }
   }
 
   if (pred) {
     #pragma unroll
     for (int i = 16; i > 0; i /= 2)
       res += __shfl_down_sync(0xffffffff, res, i);
     if (threadIdx.x % 32 == 0)
       reinterpret_cast<__half*>(C)[c_gl_wr] = __float2half(res);
   }
 }
 
 inline int ceildiv(int a, int b) {
   return (a + b - 1) / b;
 }
 
 const int THREAD_M = 16;
 
 void  code1x16_matvec_cuda(
   const void* __restrict__ A,
   const void* __restrict__ B,
         void* __restrict__ C,
   const void* __restrict__ codebook,
   int prob_m,
   int prob_k,
   const int4 codebook_a_sizes,
   const int codebook_stride
 ) {
   int sms;
   hipDeviceGetAttribute(&sms, hipDeviceAttributeMultiprocessorCount, 0);
   int waves = 0;
   int thread_m;
   do {
     waves++;
     thread_m = ceildiv(prob_m, waves * sms);
   } while (thread_m > THREAD_M);
 
   int blocks = ceildiv(prob_m, thread_m);
   int threads = 32 * thread_m;
   hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
   Code1x16MatVec<<<blocks, threads, 16*32*9, stream>>>(
     (const int4*) A,
     (const int4*) B,
     (int4*) C,
     (const int4*) codebook,
     prob_m,
     prob_k,
     codebook_a_sizes,
     codebook_stride
   );
 }
 
 void  code2x8_matvec_cuda(
   const void* __restrict__ A,
   const void* __restrict__ B,
         void* __restrict__ C,
   const void* __restrict__ codebook,
   int prob_m,
   int prob_k,
   const int4 codebook_a_sizes,
   const int codebook_stride
 ) {
   int sms;
   hipDeviceGetAttribute(&sms, hipDeviceAttributeMultiprocessorCount, 0);
   int waves = 0;
   int thread_m;
   do {
     waves++;
     thread_m = ceildiv(prob_m, waves * sms);
   } while (thread_m > THREAD_M);
 
   int blocks = ceildiv(prob_m, thread_m);
   int threads = 32 * thread_m;
   int shared = 16 * (2 * 256 * 8 + 32 * 9);
   hipFuncSetAttribute(reinterpret_cast<const void*>(
     Code2x8MatVec), hipFuncAttributeMaxDynamicSharedMemorySize, shared
   );
   hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
   Code2x8MatVec<<<blocks, threads, shared, stream>>>(
     (const int4*) A,
     (const int4*) B,
     (int4*) C,
     (const int4*) codebook,
     prob_m,
     prob_k,
     codebook_a_sizes,
     codebook_stride
   );
 }