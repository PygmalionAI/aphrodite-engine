#include "hip/hip_runtime.h"
#include <torch/all.h>
#include <torch/python.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include "cu_compat.cuh"

const int BLOCKWIDTH  = 256;
const int BLOCKHEIGHT =  32;

__device__ inline unsigned int as_unsigned(int i) {
  return *reinterpret_cast<unsigned int*>(&i);
}

__device__ inline int as_int(int i) {
  return *reinterpret_cast<int*>(&i);
}

template <typename scalar_t>
__global__ void VecQuant4MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const       int* __restrict__ zeros,
    const   	int* __restrict__ g_idx,
    int batch,
    int vec_height,
    int height,
    int width,
	int zero_width
) {
    int h = BLOCKHEIGHT * blockIdx.x;
    int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;
    int h_end = min(h + BLOCKHEIGHT, height);

    __shared__ scalar_t blockvec[BLOCKWIDTH];
    int i = width * h + w;
    int g_h = h * 8;
    int h_range = (h_end - h) * 8;
    int k;
    unsigned int g;
    scalar_t w_tmp;


    int z_w = w / 8;
    int z_mod = (w % 8) * 4;

    float weight[BLOCKWIDTH];

    if (w < width) {
        for (k = 0; k < h_range; ++k) {
    	      int k_w = (k / 8);
	          int k_bit = (k % 8) * 4;

            g = as_int(g_idx[g_h + k]);
            scalar_t scale = scales[g * width + w];
            scalar_t zero = scalar_t(((as_unsigned(zeros[g * zero_width + z_w]) >> z_mod) & 0xF) + 1);
            w_tmp = ((as_unsigned(mat[i + (k_w * width)]) >> k_bit) & 0xF);
            weight[k] = scale * (w_tmp - zero);
        }
    }

    scalar_t res;
    for (int b = 0; b < batch; ++b) {
	    res = 0;

        if (threadIdx.x < h_range) {
            blockvec[threadIdx.x] = vec[b * vec_height + blockIdx.x * BLOCKWIDTH + threadIdx.x];
        }
        __syncthreads();
        if (w < width) {
	        for (k = 0; k < h_range; ++k){
	            res += weight[k] * blockvec[k];
            }
            atomicAdd(&mul[b * width + w], res);
        }
        __syncthreads();
    }
}

void vecquant4matmul_cuda(
    torch::Tensor vec,
    torch::Tensor mat,
    torch::Tensor mul,
    torch::Tensor scales,
    torch::Tensor zeros,
    torch::Tensor g_idx
) {
    int batch = vec.size(0);
    int vec_height = vec.size(1);
    int height = mat.size(0);
    int width = mat.size(1);
    int zero_width = zeros.size(1);

    dim3 blocks(
        (height + BLOCKHEIGHT - 1) / BLOCKHEIGHT,
        (width + BLOCKWIDTH - 1) / BLOCKWIDTH
    );
    dim3 threads(BLOCKWIDTH);

    AT_DISPATCH_FLOATING_TYPES(
        vec.type(), "vecquant4matmul_cuda", ([&] {
            VecQuant4MatMulKernel<<<blocks, threads>>>(
                vec.data<scalar_t>(), mat.data<int>(), mul.data<scalar_t>(),
                scales.data<scalar_t>(), zeros.data<int>(), g_idx.data<int>(),
                batch, vec_height, height, width, zero_width
            );
        })
    );
}