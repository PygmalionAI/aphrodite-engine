#include "hip/hip_runtime.h"
#include "q_gemm.cuh"
#include "matrix_view.cuh"

#include "qdq_4.cuh"

#define BLOCK_KN_SIZE 128
#define BLOCK_M_SIZE_MAX 8
#define MAX_GROUPS_IN_BLOCK (BLOCK_KN_SIZE / 32)
#define CLEAR_N_SIZE 256
#define MAX_Q_GEMM_ROWS 50
#define DIVIDE(x, size) (((x) + (size) - 1) / (size))

#include "q_gemm_kernel_gptq.cuh"

#if defined(USE_ROCM)
__host__ __forceinline__ hipblasStatus_t __compat_hipblasHgemm(hipblasHandle_t    handle,
                                                               hipblasOperation_t transA,
                                                               hipblasOperation_t transB,
                                                               int                m,
                                                               int                n,
                                                               int                k,
                                                               const half*        alpha,
                                                               const half*        AP,
                                                               int                lda,
                                                               const half*        BP,
                                                               int                ldb,
                                                               const half*        beta,
                                                               half*              CP,
                                                               int                ldc) {
    return hipblasHgemm(handle, transA, transB, m, n, k,
                        reinterpret_cast<const hipblasHalf *>(alpha),
                        reinterpret_cast<const hipblasHalf *>(AP), lda,
                        reinterpret_cast<const hipblasHalf *>(BP), ldb,
                        reinterpret_cast<const hipblasHalf *>(beta),
                        reinterpret_cast<hipblasHalf *>(CP), ldc);
}
#define hipblasHgemm __compat_hipblasHgemm

// Previous version of PyTorch were converting to rocBLAS instead of hipBLAS.
#define rocblas_operation_none HIPBLAS_OP_N
#define rocblas_hgemm __compat_hipblasHgemm
#endif

void gemm_half_q_half_cuda_part
(
    const half* a,
    QMatrix* b,
    half* c,
    int size_m,
    int size_n,
    int size_k,
    int m_count,
    bool clear
)
{
    dim3 blockDim, gridDim;
    blockDim.x = BLOCK_KN_SIZE;
    blockDim.y = 1;
    blockDim.z = 1;
    gridDim.x = DIVIDE(size_n, BLOCK_KN_SIZE * 4);
    gridDim.y = DIVIDE(size_m, m_count);
    gridDim.z = DIVIDE(size_k, BLOCK_KN_SIZE);

    fp_gemm_half_q_half_gptq_kernel kernel = pick_gemm_half_q_half_gptq_kernel(true, m_count);

    kernel<<<gridDim, blockDim>>>
    (
        a,
        b->cuda_q_weight,
        b->cuda_gptq_qzeros,
        b->cuda_gptq_scales,
        c,
        size_m,
        size_n,
        size_k,
        b->groups,
        b->groupsize,
        b->cuda_q_perm,
        clear
    );
}

void gemm_half_q_half_cuda
(
    hipblasHandle_t cublas_handle,
    const half* a,
    QMatrix* b,
    half* c,
    int size_m,
    int size_n,
    int size_k,
    bool clear,
    half* temp_dq,
    bool force_cuda
)
{
    if (size_m > MAX_Q_GEMM_ROWS && !force_cuda)
    {

        // Reconstruct FP16 matrix, then cuBLAS

        if (!temp_dq) temp_dq = b->temp_dq;
        b->reconstruct(temp_dq);

        //hipblasSetMathMode(cublas_handle, HIPBLAS_TENSOR_OP_MATH);

        const half alpha = __float2half(1.0f);
        const half beta = clear ? __float2half(0.0f) : __float2half(1.0f);
        hipblasHgemm(cublas_handle,
                    HIPBLAS_OP_N,
                    HIPBLAS_OP_N,
                    size_n, size_m, size_k,
                    &alpha, temp_dq, size_n,
                            a,       size_k,
                    &beta,  c,       size_n);

    }
    else
    {
        // Quantized matmul

        //if (clear) clear_tensor_cuda(c, size_m, size_n);

        int max_chunks = size_m / BLOCK_M_SIZE_MAX;
        int last_chunk = max_chunks * BLOCK_M_SIZE_MAX;
        int last_chunk_size = size_m - last_chunk;

        if (max_chunks)
        {
            gemm_half_q_half_cuda_part(a, b, c, last_chunk, size_n, size_k, BLOCK_M_SIZE_MAX, clear);
        }

        if (last_chunk_size)
        {
            gemm_half_q_half_cuda_part(a + last_chunk * size_k, b, c + last_chunk * size_n, last_chunk_size, size_n, size_k, last_chunk_size, clear);
        }
    }
}

__global__ void clear_kernel
(
    half* __restrict__ c,
    const int size_m,
    const int size_n
)
{
    int m = blockIdx.y;
    int n = (blockIdx.x * CLEAR_N_SIZE + threadIdx.x) * 8;
    if (n >= size_n) return;
    int4* c_ptr = (int4*)(c + m * size_n + n);
    *c_ptr = {};
}

void clear_tensor_cuda
(
    half* c,
    int size_m,
    int size_n
)
{
    return;
    dim3 blockDim, gridDim;
    blockDim.x = CLEAR_N_SIZE;
    blockDim.y = 1;
    gridDim.x = DIVIDE(size_n / 8, CLEAR_N_SIZE);
    gridDim.y = size_m;
    clear_kernel<<<gridDim, blockDim>>>(c, size_m, size_n);
}