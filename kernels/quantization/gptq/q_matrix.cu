#include "hip/hip_runtime.h"
#include "q_matrix.cuh"
#include "matrix_view.cuh"

#include "qdq_4.cuh"

#define BLOCK_KN_SIZE 128

#define THREADS_X 32
#define THREADS_Y 32
#define DIVIDE(x, size) (((x) + (size) - 1) / (size))

// Shuffle quantized data on load

__global__ void shuffle_kernel
(
    uint32_t* __restrict__ b_q_weight,
    const int size_k,
    const int size_n
)
{
    int n = blockIdx.x * THREADS_X + threadIdx.x;
    if (n >= size_n) return;
    int k = 0;
    uint32_t* b_ptr = b_q_weight + n;
    while (k < size_k) { shuffle_4bit_8 (b_ptr, size_n); b_ptr += 1 * size_n; k +=  8; }
}


// QMatrix constructor

QMatrix::QMatrix
(
    const int _device,
    const int _height,
    const int _width,
    const int _groups,

    uint32_t* _q_weight,
    uint16_t* _q_perm,
    uint16_t* _q_invperm,

    uint32_t* _gptq_qzeros,
    half* _gptq_scales,
    uint32_t* _gptq_g_idx,

    half* _temp_dq
) :
    device(_device),
    height(_height),
    width(_width),
    groups(_groups),
    temp_dq(_temp_dq)
{
    hipSetDevice(device);

    cuda_q_weight = _q_weight;
    cuda_q_perm = _q_perm;
    cuda_q_invperm = _q_invperm;
    cuda_gptq_qzeros = _gptq_qzeros;
    cuda_gptq_scales = _gptq_scales;

    is_gptq = true;

    groupsize = 1;
    while (groupsize * groups < height) groupsize *= 2;

    if (_gptq_g_idx) make_sequential(_gptq_g_idx);

    // Shuffle quantized data

    dim3 blockDim, gridDim;
    blockDim.x = THREADS_X;
    blockDim.y = 1;
    gridDim.x = DIVIDE(width, THREADS_X);
    gridDim.y = 1;

    shuffle_kernel<<<gridDim, blockDim>>>(cuda_q_weight, height, width);
}


// Reconstruct b[k,n] (GPTQ)

__global__ void reconstruct_gptq_kernel
(
    const uint32_t* __restrict__ b_q_weight,
    const uint16_t* __restrict__ b_q_perm,
    const uint32_t* __restrict__ b_gptq_qzeros,
    const half* __restrict__ b_gptq_scales,
    const int size_k,
    const int size_n,
    const int groupsize,
    const int groups,
    half* __restrict__ b
)
{
    MatrixView_half_rw b_(b, size_k, size_n);
    MatrixView_q4_row b_gptq_qzeros_(b_gptq_qzeros, groups, size_n);
    MatrixView_half b_gptq_scales_(b_gptq_scales, groups, size_n);

    int offset_k = BLOCK_KN_SIZE * blockIdx.y;
    int offset_n = BLOCK_KN_SIZE * blockIdx.x * 4;

    int end_k = min(offset_k + BLOCK_KN_SIZE, size_k);

    // Preload remapping table

    __shared__ uint16_t perm[BLOCK_KN_SIZE];
    int t = threadIdx.x;

    if (b_q_perm)
    {
        if (offset_k + t < size_k)
            perm[t] = b_q_perm[offset_k + t];
    }

    // Column

    int n = offset_n + t * 4;
    if (n >= size_n) return;

    // Find initial group

    int group = offset_k / groupsize;
    int nextgroup = offset_k + groupsize;

    // b offset

    int qk = offset_k / (32 / 4);

    const uint32_t* b_ptr = b_q_weight + qk * size_n + n;

    // Initial zeros/scale

    int zeros[4];
    half2 scales[4];
    half2 z1z16[4][2];
    half2 y1y16[4][2];
    b_gptq_qzeros_.item4(zeros, group, n);
    b_gptq_scales_.item4_h2(scales, group, n);
    dequant_4bit_8_prep_zero(zeros[0] + 1, z1z16[0], y1y16[0]);
    dequant_4bit_8_prep_zero(zeros[1] + 1, z1z16[1], y1y16[1]);
    dequant_4bit_8_prep_zero(zeros[2] + 1, z1z16[2], y1y16[2]);
    dequant_4bit_8_prep_zero(zeros[3] + 1, z1z16[3], y1y16[3]);

    __syncthreads();

    int k = offset_k;
    int lk = 0;

    while (k < end_k)
    {
        if (k == nextgroup)
        {
            group++;
            nextgroup += groupsize;
            b_gptq_qzeros_.item4(zeros, group, n);
            b_gptq_scales_.item4_h2(scales, group, n);
            dequant_4bit_8_prep_zero(zeros[0] + 1, z1z16[0], y1y16[0]);
            dequant_4bit_8_prep_zero(zeros[1] + 1, z1z16[1], y1y16[1]);
            dequant_4bit_8_prep_zero(zeros[2] + 1, z1z16[2], y1y16[2]);
            dequant_4bit_8_prep_zero(zeros[3] + 1, z1z16[3], y1y16[3]);
        }

        for (int p = 0; p < 4; p++)
        {
            half2 dq[4][4];
            const int4* b_ptr4 = (int4*) b_ptr;
            int4 load_int4 = *b_ptr4;

            dequant_4bit_8_gptq(load_int4.x, dq[0], z1z16[0], y1y16[0], size_n, false);
            dequant_4bit_8_gptq(load_int4.y, dq[1], z1z16[1], y1y16[1], size_n, false);
            dequant_4bit_8_gptq(load_int4.z, dq[2], z1z16[2], y1y16[2], size_n, false);
            dequant_4bit_8_gptq(load_int4.w, dq[3], z1z16[3], y1y16[3], size_n, false);

            b_ptr += size_n;
            //half* dqh = (half*)dq;
            if (b_q_perm)
            {
                for (int j = 0; j < 4; j++)
                {
                    for (int v = 0; v < 4; v++) dq[v][j] = __hmul2(scales[v], dq[v][j]);
                    b_.set4(perm[lk++], n, __low2half(dq[0][j]), __low2half(dq[1][j]), __low2half(dq[2][j]), __low2half(dq[3][j]));
                    b_.set4(perm[lk++], n, __high2half(dq[0][j]), __high2half(dq[1][j]), __high2half(dq[2][j]), __high2half(dq[3][j]));
                }
            }
            else
            {
                for (int j = 0; j < 4; j++)
                {
                    for (int v = 0; v < 4; v++) dq[v][j] = __hmul2(scales[v], dq[v][j]);
                    b_.set4(offset_k + lk++, n, __low2half(dq[0][j]), __low2half(dq[1][j]), __low2half(dq[2][j]), __low2half(dq[3][j]));
                    b_.set4(offset_k + lk++, n, __high2half(dq[0][j]), __high2half(dq[1][j]), __high2half(dq[2][j]), __high2half(dq[3][j]));
                }
            }
        }
        k += 32;
    }
}

void QMatrix::reconstruct(half* out)
{
    dim3 blockDim, gridDim;
    blockDim.x = BLOCK_KN_SIZE;
    blockDim.y = 1;
    gridDim.x = DIVIDE(width, BLOCK_KN_SIZE);
    gridDim.y = DIVIDE(height, BLOCK_KN_SIZE);

    reconstruct_gptq_kernel<<<gridDim, blockDim>>>
    (
        cuda_q_weight,
        cuda_q_perm,
        cuda_gptq_qzeros,
        cuda_gptq_scales,
        height,
        width,
        groupsize,
        groups,
        out
    );
}

__global__ void make_sequential_kernel
(
    const uint32_t* __restrict__ w,
    uint32_t* __restrict__ w_new,
    const uint16_t* __restrict__ q_perm,
    const int w_height,
    const int w_width
)
{
    const uint64_t* w2 = (uint64_t*) w;
    uint64_t* w_new2 = (uint64_t*) w_new;
    int w2_stride = w_width >> 1;

    int w2_column = THREADS_X * blockIdx.x + threadIdx.x;
    if (w2_column >= w2_stride) return;

    int w_new2_row = blockIdx.y;

    int q_perm_idx = w_new2_row << 3;

    uint64_t dst = 0;

    #pragma unroll
    for (int i = 0; i < 8; i++)
    {
        int source_row = q_perm[q_perm_idx++];

        int w2_row = source_row >> 3;
        int w2_subrow = source_row & 0x07;
        int w2_row_shift = w2_subrow << 2;
        int wnew2_row_shift = i << 2;

        uint64_t src = w2[w2_row * w2_stride + w2_column];
        src >>= w2_row_shift;
        src &= 0x0000000f0000000f;
        src <<= wnew2_row_shift;
        dst |= src;
    }

    w_new2[w_new2_row * w2_stride + w2_column] = dst;
}

void QMatrix::make_sequential(const uint32_t* cpu_g_idx)
{
    uint32_t* cuda_new_qweight = NULL;
    hipMalloc(&cuda_new_qweight, height / 8 * width * sizeof(uint32_t));

    uint32_t* cpu_g_idx_map = (uint32_t*) calloc(groups, sizeof(uint32_t));
    uint32_t* cpu_x_map = (uint32_t*) malloc(height * sizeof(uint32_t));
    uint32_t* cpu_x_map_inv = (uint32_t*) malloc(height * sizeof(uint32_t));

    // Group histogram

    for (int i = 0; i < height; i++) cpu_g_idx_map[cpu_g_idx[i]]++;

    // Group map

    for (int i = 0, acc = 0; i < groups; i++)
    {
        short tmp = cpu_g_idx_map[i];
        cpu_g_idx_map[i] = acc;
        acc += tmp;
    }

    // X map (inverse)

    for (int row = 0; row < height; row++)
    {
        uint32_t target_group = cpu_g_idx[row];
        uint32_t target_row = cpu_g_idx_map[target_group];
        cpu_g_idx_map[target_group]++;
        cpu_x_map_inv[row] = target_row;
    }

    // X map

    for (int row = 0; row < height; row++) cpu_x_map[cpu_x_map_inv[row]] = row;

    // Reduce to uint16_t

    uint16_t* cpu_x_map16 = (uint16_t*)cpu_x_map;
    uint16_t* cpu_x_map_inv16 = (uint16_t*)cpu_x_map_inv;
    for (int row = 0; row < height; row++) cpu_x_map16[row] = (uint16_t) cpu_x_map[row];
    for (int row = 0; row < height; row++) cpu_x_map_inv16[row] = (uint16_t) cpu_x_map_inv[row];

    // Move to CUDA

    hipMemcpyAsync(cuda_q_perm, cpu_x_map16, height * sizeof(uint16_t), hipMemcpyHostToDevice);
    hipMemcpyAsync(cuda_q_invperm, cpu_x_map_inv16, height * sizeof(uint16_t), hipMemcpyHostToDevice);

    // Rearrange rows in w

    dim3 blockDim, gridDim;
    blockDim.x = THREADS_X;
    blockDim.y = 1;
    gridDim.x = DIVIDE(width, THREADS_X);
    gridDim.y = height / 8;

    make_sequential_kernel<<<gridDim, blockDim>>>
    (
        cuda_q_weight,
        cuda_new_qweight,
        cuda_q_perm,
        height / 8,
        width
    );

    // Replace qweights

    hipMemcpyAsync(cuda_q_weight, cuda_new_qweight, height / 8 * width * sizeof(uint32_t), hipMemcpyDeviceToDevice);

    // Cleanup

    hipDeviceSynchronize();

    hipFree(cuda_new_qweight);
    free(cpu_g_idx_map);
    free(cpu_x_map);
    free(cpu_x_map_inv);
}