#include "hip/hip_runtime.h"
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 700
  #include <mma.h>
#endif

#include <ATen/ATen.h>
#include <ATen/core/Tensor.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/DeviceGuard.h>
#include <torch/extension.h>
#include <c10/cuda/CUDAGuard.h>


template <typename U, typename V>
constexpr __host__ __device__ auto divDown(U a, V b) -> decltype(a + b) {
  static_assert(std::is_integral<U>::value && std::is_integral<V>::value, "");
  return (a / b);
}

template <typename U, typename V>
constexpr __host__ __device__ auto divUp(U a, V b) -> decltype(a + b) {
  static_assert(std::is_integral<U>::value && std::is_integral<V>::value, "");
  // Overflow safe variant of (a + b - 1) / b
  const uint64_t blocks = a / b + (a % b != 0);
  return blocks;
}

template <typename U, typename V>
constexpr __host__ __device__ auto roundDown(U a, V b) -> decltype(a + b) {
  static_assert(std::is_integral<U>::value && std::is_integral<V>::value, "");
  return divDown(a, b) * b;
}

template <typename U, typename V>
constexpr __host__ __device__ auto roundUp(U a, V b) -> decltype(a + b) {
  static_assert(std::is_integral<U>::value && std::is_integral<V>::value, "");
  return divUp(a, b) * b;
}

constexpr int32_t kWarpSize = 32;
constexpr int32_t KTilesPerWarp = 8;
constexpr int32_t kMTileSize = 16;
constexpr int32_t kNTileSize = 8;
constexpr int32_t kKTileSize = 16;

struct __align__(16) f16x2x4_u32 {
  uint32_t vals[4];
};
struct __align__(16) f16x2x2_u32 {
  uint32_t vals[2];
};

struct ALayout_RM {
template <int KTilesToLoad>
static __device__ void load(
    const half* A,
    int32_t m,
    int32_t k,
    int32_t mTiles,
    int32_t mTile,
    int32_t kTiles,
    int32_t kTileStart,
    int32_t laneId,
    f16x2x4_u32 out[KTilesToLoad]) {
  const auto mLane = mTile * kMTileSize + (laneId / 4);
  const auto kLane = kTileStart * kKTileSize + (laneId % 4) * 4;

  // access
  // [mTile * kMTileSize + (laneId / 4)]
  // [kTileStart * kKTileSize + (laneId % 4) * 2]
  auto aPtr = A + mLane * k + kLane;

  auto aPtrPlus8Rows = aPtr + 8 * k;

  bool m0InBounds = mLane < m;
  bool m1InBounds = (mLane + 8) < m;

#pragma unroll
  for (int i = 0; i < KTilesToLoad; ++i) {
    out[i].vals[0] = m0InBounds
          ? *reinterpret_cast<const uint32_t*>(aPtr  + i * kKTileSize)
          : uint32_t(0);
    out[i].vals[1] = m1InBounds
          ? *reinterpret_cast<const uint32_t*>(aPtrPlus8Rows  + i * kKTileSize)
          : uint32_t(0);

    out[i].vals[2] = m0InBounds
          ? *reinterpret_cast<const uint32_t*>(aPtr  + i * kKTileSize + 2)
          : uint32_t(0);
    out[i].vals[3] = m1InBounds ? *reinterpret_cast<const uint32_t*>(
                                        aPtrPlus8Rows  + i * kKTileSize + 2)
                                  : uint32_t(0);
  }
}

static __device__ void store(
    half* C,
    int32_t m,
    int32_t n,
    int32_t mOutTiles,
    int32_t mTile,
    int32_t nOutTiles,
    int32_t nTile,
    int32_t laneId,
    const float4& out) {
#if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 800

  // sum.x / sum.y are written at
  // [laneId / 4], [(laneId % 4) * 2, (laneId % 4) * 2 + 1]
  // sum.z / sum.w are written at
  // [8 + (laneId / 4)], [(laneId % 4) * 2, (laneId % 4) * 2 + 1]
  // i.e., same columns, different row.
  const int outRow = mTile * kMTileSize + (laneId / 4);
  const int outCol = nTile * kNTileSize + (laneId % 4) * 2;

  // Pointer where sum.x / sum.y is written
  auto cPtr = C + outRow * n + outCol;

  auto v01 = __float22half2_rn(float2{out.x, out.y});
  auto v23 = __float22half2_rn(float2{out.z, out.w});

  if (outRow < m) {
    *reinterpret_cast<half2*>(cPtr) = v01;
  }

  // sum.z, sum.w at +8 rows from cPtr
  if (outRow + 8 < m) {
    *reinterpret_cast<half2*>(cPtr + 8 * n) = v23;
  }
#endif
}
};

struct BLayout_D4 {
static constexpr bool use_codebook = true;

template <int KTilesPerIteration>
static __device__ void load(
    const void* __restrict__ B,
    const uint64_t* __restrict__ CB,
    int32_t n,
    int32_t k,
    int32_t nTiles,
    int32_t nTile,
    int32_t kTiles,
    int32_t kTileStart,
    int32_t laneId,
    f16x2x2_u32 b[KTilesPerIteration]) {
#if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 800
  auto Bptr = reinterpret_cast<const uint8_t*>(B);
  #pragma unroll
  for (int i = 0; i < KTilesPerIteration; ++i) {
       const int row = nTile * kNTileSize + laneId / 4;
       const int col = (kTileStart + i) * kKTileSize / 4 + laneId % 4;
       *(reinterpret_cast<uint64_t*>(b[i].vals)) = CB[Bptr[row * k/4 + col]];
  }
#endif
}
};

struct BLayout_HI {
static constexpr bool use_codebook = false;

template <int KTilesPerIteration>
static __device__ void load(
    const void* __restrict__ B,
    const uint64_t* __restrict__ CB,
    int32_t n,
    int32_t k,
    int32_t nTiles,
    int32_t nTile,
    int32_t kTiles,
    int32_t kTileStart,
    int32_t laneId,
    f16x2x2_u32 b[KTilesPerIteration]) {
#if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 800
  auto Bptr = reinterpret_cast<const uint32_t*>(B);
  #pragma unroll
  for (int i = 0; i < KTilesPerIteration; ++i) {
      const int row = nTile * kNTileSize + laneId / 4;
      const int col = (kTileStart + i) * kKTileSize / 8 + (laneId % 4) / 2;
      // simply use code - 7.5 instead of reading codebook
      uint32_t code = Bptr[row * k/8 + col];

      const uint32_t c0 = 0x64086408;
      const half y16_ = __float2half_rn(1.0f / 16.0f);
      const half2 y16 = __halves2half2(y16_, y16_);
      const half z16_ = __float2half_rn(-1024.0f / 16.0f - 8.0f);
      const half2 z16 = __halves2half2(z16_, z16_);

      uint32_t qa = code >> ((laneId & 1) * 8);
      uint32_t q0 = (((qa & 0x000f000f) << 4)| c0);
      uint32_t q1 = ((qa & 0x00f000f0) | c0);
      *(half2*)(b[i].vals) = __hfma2(*((half2*)(&q0)), y16, z16);
      *(half2*)(b[i].vals+1) = __hfma2(*((half2*)(&q1)), y16, z16);
  }
#endif
}
};

struct BLayout_E8 {
static constexpr bool use_codebook = true;

__device__ static inline uint64_t decode8weights(
    uint16_t weight_compressed,
    const int64_t *__restrict__ codebook_abs
) {
#if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 800

    uint8_t bits_sign = weight_compressed & 0xff;
    uint8_t parity = __popc(bits_sign) & 1;
    uint8_t sign_vec = bits_sign ^ parity;
    uint8_t bits_abs = (weight_compressed >> 8);
    int64_t packed = codebook_abs[bits_abs];

    uint64_t decoded_sign = sign_vec * 0x8040201008040201ll;
    decoded_sign &= 0x8080808080808080;
    decoded_sign >>= 7;
    decoded_sign *= 255 - 3;
    packed ^= decoded_sign;
    packed |= 0x0101010101010101;
    packed -= parity * 0x0202020202020202;

    return packed;
#endif
}

__device__ static inline uint32_t decode8weights(
    uint16_t weight_compressed,
    const int64_t *__restrict__ codebook_abs,
    int idx
) {
#if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 800
    uint8_t bits_sign = weight_compressed & 0xff; //__brev(weight_compressed) >> 24;
    const uint32_t magic_nums[2] = {0x08040201ll, 0x80402010ll};
    uint8_t parity = __popc(bits_sign) & 1;
    uint8_t sign_vec = bits_sign ^ parity; // (parity << 7);
    uint16_t bits_abs = (weight_compressed >> 8);
    uint32_t packed = ((uint32_t*)codebook_abs)[(bits_abs << 1) + idx];
    uint32_t magic_num = magic_nums[idx];
    uint32_t decoded_sign = sign_vec * magic_num;
    decoded_sign &= 0x80808080;
    decoded_sign >>= 7;
    decoded_sign *= 255 - 3;
    packed ^= decoded_sign;
    packed |= 0x01010101;
    packed -= parity * 0x02020202;
    return packed;
#endif
};

template <int KTilesPerIteration>
static __device__ void load(
    const void* __restrict__ B,
    const uint64_t* __restrict__ CB,
    int32_t n,
    int32_t k,
    int32_t nTiles,
    int32_t nTile,
    int32_t kTiles,
    int32_t kTileStart,
    int32_t laneId,
    f16x2x2_u32 b[KTilesPerIteration]) {
#if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 800
  auto Bptr = (const uint16_t*) B;
  #pragma unroll
  for (int i = 0; i < KTilesPerIteration; ++i) {
       const int row = nTile * kNTileSize + laneId / 4;
       const int col = (kTileStart + i) * kKTileSize / 8 + laneId % 4 / 2;
       uint32_t decoded = decode8weights(Bptr[row * k/8 + col], (const int64_t*)CB, laneId & 1);
       half2 unpacked[2];
       uint32_t lower_half = decoded & 0x00ff00ff;
       lower_half = (lower_half ^ 0x5c805c80);
       memcpy(unpacked, &lower_half, sizeof(uint32_t));
       uint32_t upper_half = (decoded & 0xff00ff00) >> 8;
       upper_half = (upper_half ^ 0x5c805c80);
       memcpy(unpacked + 1, &upper_half, sizeof(uint32_t));

       const half adjust_ = __float2half_rn(-288.0f);
       const half2 adjust = __halves2half2(adjust_, adjust_);
       unpacked[0] = __hadd2(unpacked[0], adjust);
       unpacked[1] = __hadd2(unpacked[1], adjust);
       *(reinterpret_cast<uint64_t*>(b[i].vals)) = *(reinterpret_cast<uint64_t*>(unpacked));
       //*((half*)(b[i].vals)) = unpacked[0];
       //*((half*)(b[i].vals) + 1) = unpacked[0].y;
       //*((half*)(b[i].vals) + 2) = unpacked[1].x;
       //*((half*)(b[i].vals) + 3) = unpacked[1].y;
  }
#endif
}
};


template <
    typename ALayout,
    typename BLayout,
    typename CLayout,
    int Warps,
    int KTilesPerIteration>
__global__
__launch_bounds__(256) void tinygemm_m16n8k16_chunk_kernel(
    // Data for the A matrix, loaded as per ALayout
    const half* __restrict__ A,
    const void* __restrict__ B,
    const uint64_t* __restrict__ CB,

    // Output data for the C matrix, stored as per CLayout
    half* __restrict__ C,

    // The size of the matrix multiplication
    int32_t m,
    int32_t n,
    int32_t k,

    // The size of the matrix multiplication, in multiples of our TC tile size
    int32_t mTiles,
    int32_t nTiles,
    int32_t kTiles) {
#if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 800
  __shared__ uint64_t CB_[256];
  if (BLayout::use_codebook) {
    CB_[threadIdx.x + threadIdx.y * 32] = CB[threadIdx.x + threadIdx.y * 32];
    __syncthreads();
  }

  auto warpId = threadIdx.y;
  auto laneId = threadIdx.x;

  int32_t mTile = blockIdx.z;
  int32_t nTile = blockIdx.y;

  float4 c{0.0f, 0.0f, 0.0f, 0.0f};

 // First, handle whole multiples of KTilesPerIteration
  auto kTilesLimit = roundDown(kTiles, KTilesPerIteration);

  // Each warp handles a set of KTilesPerIteration under the above limit
  for (int32_t kTileBase = warpId * KTilesPerIteration; kTileBase < kTilesLimit; kTileBase += Warps * KTilesPerIteration) {
    //
    // Load data from A
    //
    f16x2x4_u32 a[KTilesPerIteration];
    ALayout::template load<KTilesPerIteration>(
        A, m, k, mTiles, mTile, kTiles, kTileBase, laneId, a);

    //
    // Load data from B and de-quantize as needed
    //
    f16x2x2_u32 b[KTilesPerIteration];
    BLayout::template load<KTilesPerIteration>(
        B, CB_, n, k, nTiles, nTile, kTiles, kTileBase, laneId, b);

    // Now, perform the matrix multiplication
    //
    #pragma unroll
    for (int i = 0; i < KTilesPerIteration / 2; ++i) {
      float4 cTmp[2];

      #pragma unroll
      for (int k = 0; k < 2; ++k) {
        cTmp[k] = float4{0.0f, 0.0f, 0.0f, 0.0f};
      }

      #pragma unroll
      for (int k = 0; k < 2; ++k) {
        asm volatile(
              "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32 "
              "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};"
              : "=f"(cTmp[k].x),
                "=f"(cTmp[k].y),
                "=f"(cTmp[k].z),
                "=f"(cTmp[k].w)
              : "r"(a[i * 2 + k].vals[0]),
                "r"(a[i * 2 + k].vals[1]),
                "r"(a[i * 2 + k].vals[2]),
                "r"(a[i * 2 + k].vals[3]),
                "r"(b[i * 2 + k].vals[0]),
                "r"(b[i * 2 + k].vals[1]),
                "f"(cTmp[k].x),
                "f"(cTmp[k].y),
                "f"(cTmp[k].z),
                "f"(cTmp[k].w));
      }
      #pragma unroll
      for (int k = 0; k < 2; ++k) {
        c.x += cTmp[k].x;
        c.y += cTmp[k].y;
        c.z += cTmp[k].z;
        c.w += cTmp[k].w;
      }
    }

  } // for all tiles under kTilesLimit


  auto kTileBaseRemaining = kTilesLimit + warpId;

  // If we have any remainder k-tiles, some warps will handle them, processing
  // kInnerKTiles k-tiles at a time
  if (kTileBaseRemaining < kTiles) {
    f16x2x4_u32 a;
    ALayout::template load<1>(
        A, m, k, mTiles, mTile, kTiles, kTileBaseRemaining, laneId, &a);

    f16x2x2_u32 b;
    BLayout::template load<1>(
        B, CB, n, k, nTiles, nTile, kTiles, kTileBaseRemaining, laneId, &b);

    asm volatile(
              "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32 "
              "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};"
              : "=f"(c.x),
                "=f"(c.y),
                "=f"(c.z),
                "=f"(c.w)
              : "r"(a.vals[0]),
                "r"(a.vals[1]),
                "r"(a.vals[2]),
                "r"(a.vals[3]),
                "r"(b.vals[0]),
                "r"(b.vals[1]),
                "f"(c.x),
                "f"(c.y),
                "f"(c.z),
                "f"(c.w));
  }
  // Reduce independent k-tiles (same m/n) across warps
  __shared__ float4 smem_sum[Warps][kWarpSize];

  smem_sum[warpId][laneId] = c;

  __syncthreads();

  if (warpId == 0) {
    float4 sum_f32{0.0f, 0.0f, 0.0f, 0.0f};

    // Reduce across the block in the first warp
    for (int i = 0; i < Warps; ++i) {
      float4 v = smem_sum[i][laneId];
      sum_f32.x += v.x;
      sum_f32.y += v.y;
      sum_f32.z += v.z;
      sum_f32.w += v.w;
    }

    // Write the reduced result (in the first warp) into the output
    CLayout::store(
        C,
        m,
        n,
        mTiles,
        mTile,
        // n for C output becomes k for A input, so for m16n8k16,
        // we need to halve the tiles
        nTiles / 2,
        nTile,
        laneId,
        sum_f32);
  }
#endif
}

at::Tensor d4_mm_origorder(
    const at::Tensor& A,
    const at::Tensor& B,
    const at::Tensor& CB) {
#if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 800
  c10::cuda::CUDAGuard g(A.device());
  auto stream = at::cuda::getCurrentCUDAStream();

  constexpr int Warps = 8;

  // row major layout
  auto m = A.size(0);
  auto mTiles = divUp(m, kMTileSize);

  // tensor core layout
  auto n = B.size(0);
  auto nTiles = divUp(n, kNTileSize);

  // row major layout
  auto k = A.size(1);
  auto kTiles = divUp(k, kKTileSize);

  // Output is a standard row-major matrix
  auto C_final = at::empty(
      {m, n}, at::TensorOptions().dtype(A.dtype()).device(A.device()));

  auto grid = dim3(1, nTiles, mTiles);
  auto block = dim3(kWarpSize, Warps);
  auto kernel = tinygemm_m16n8k16_chunk_kernel<ALayout_RM, BLayout_D4, ALayout_RM, 8, 8>;

  kernel<<<grid, block, 0, stream>>>(
      (const half*)A.data_ptr(),
      (const void*)B.data_ptr(),
      (const uint64_t*)CB.data_ptr(),
      (half*)C_final.data_ptr(),
      m,
      n,
      k,
      mTiles,
      nTiles,
      kTiles);

  return C_final;
#endif
}

at::Tensor e8p_mm_origorder(
    const at::Tensor& A,
    const at::Tensor& B,
    const at::Tensor& CB) {
#if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 800
  c10::cuda::CUDAGuard g(A.device());
  auto stream = at::cuda::getCurrentCUDAStream();

  constexpr int Warps = 8;

  // row major layout
  auto m = A.size(0);
  auto mTiles = divUp(m, kMTileSize);

  // tensor core layout
  auto n = B.size(0);
  auto nTiles = divUp(n, kNTileSize);

  // row major layout
  auto k = A.size(1);
  auto kTiles = divUp(k, kKTileSize);

  // Output is a standard row-major matrix
  auto C_final = at::empty(
      {m, n}, at::TensorOptions().dtype(A.dtype()).device(A.device()));

  auto grid = dim3(1, nTiles, mTiles);
  auto block = dim3(kWarpSize, Warps);
  auto kernel = tinygemm_m16n8k16_chunk_kernel<ALayout_RM, BLayout_E8, ALayout_RM, 8, 8>;
  kernel<<<grid, block, 0, stream>>>(
      (const half*)A.data_ptr(),
      (const void*)B.data_ptr(),
      (const uint64_t*)CB.data_ptr(),
      (half*)C_final.data_ptr(),
      m,
      n,
      k,
      mTiles,
      nTiles,
      kTiles);

  return C_final;
#endif
}

at::Tensor hi_mm_origorder(
    const at::Tensor& A,
    const at::Tensor& B) {
#if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 800
  c10::cuda::CUDAGuard g(A.device());
  auto stream = at::cuda::getCurrentCUDAStream();

  constexpr int Warps = 8;

  // row major layout
  auto m = A.size(0);
  auto mTiles = divUp(m, kMTileSize);

  // tensor core layout
  auto n = B.size(0);
  auto nTiles = divUp(n, kNTileSize);

  // row major layout
  auto k = A.size(1);
  auto kTiles = divUp(k, kKTileSize);

  // Output is a standard row-major matrix
  auto C_final = at::empty(
      {m, n}, at::TensorOptions().dtype(A.dtype()).device(A.device()));

  auto grid = dim3(1, nTiles, mTiles);
  auto block = dim3(kWarpSize, Warps);
  auto kernel = tinygemm_m16n8k16_chunk_kernel<ALayout_RM, BLayout_HI, ALayout_RM, 8, 8>;
  kernel<<<grid, block, 0, stream>>>(
      (const half*)A.data_ptr(),
      (const void*)B.data_ptr(),
      nullptr,
      (half*)C_final.data_ptr(),
      m,
      n,
      k,
      mTiles,
      nTiles,
      kTiles);

  return C_final;
#endif
}

#define DECOMPRESS_D4_BLOCK_SIZE 256

__global__ void cuda_decompress_d4_origorder_kernel(
    const uint8_t* __restrict__ YIs,	  // m x (n/4)
    const c10::Half* __restrict__ CB,           // 256 x 4
    c10::Half* __restrict__ Y             // m x n
) {
#if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 800
  const long i = threadIdx.x + DECOMPRESS_D4_BLOCK_SIZE * blockIdx.x;

  for(long r = 0; r < 4; r++) {
    uint8_t yidx = ((uint8_t*)YIs)[i*4 + r];
    ((uint64_t*)Y)[i*4 + r] = ((uint64_t*)CB)[yidx & 255];
  }
#endif
}


void decompress_d4_origorder(
    torch::Tensor YIs,      // m x (n/4)
    torch::Tensor CB,       // 256 x 4
    torch::Tensor Y         // m x n
) {
#if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 800
  size_t m = Y.sizes()[0];
  size_t n = Y.sizes()[1];

  assert(YIs.is_contiguous());
  assert(CB.is_contiguous());
  assert(Y.is_contiguous());

  assert(YIs.sizes()[0] == m);
  assert(YIs.sizes()[1] * 4 == n);
  assert(CB.sizes()[0] == 256);

  const dim3 threads(DECOMPRESS_D4_BLOCK_SIZE);
  const dim3 blocks(m*n/(16*DECOMPRESS_D4_BLOCK_SIZE));
  hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
  cuda_decompress_d4_origorder_kernel<<<blocks, threads, 0, stream>>>(
    YIs.data_ptr<uint8_t>(),
    CB.data_ptr<c10::Half>(),
    Y.data_ptr<c10::Half>()
  );
#endif
}

#define DECOMPRESS_E8P_BLOCK_SIZE 256

__global__ void cuda_decompress_e8p_origorder_kernel(
    const int16_t* __restrict__ YIs,	  // m x (n/8)
    const int64_t* __restrict__ CB, // 256 x 8
    c10::Half* __restrict__ Y             // m x n
) {
#if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 800
  const long i = threadIdx.x + DECOMPRESS_E8P_BLOCK_SIZE * blockIdx.x;
  uint16_t yidx = ((uint16_t*)YIs)[i];
  uint64_t decoded =  BLayout_E8::decode8weights(yidx, CB);

  half2 unpacked[2][2];
  uint64_t lower_half = decoded & 0x00ff00ff00ff00ff;
  lower_half = (lower_half ^ 0x5c805c805c805c80);
  memcpy(unpacked[0], &lower_half, sizeof(uint64_t));
  uint64_t upper_half = (decoded & 0xff00ff00ff00ff00) >> 8;
  upper_half = (upper_half ^ 0x5c805c805c805c80);
  memcpy(unpacked[1], &upper_half, sizeof(uint64_t));

  const half adjust_ = __float2half_rn(-288.0f);
  const half2 adjust = __halves2half2(adjust_, adjust_);

  ((__half2*)Y)[i*4] = __hadd2(unpacked[0][0], adjust); // 01
  ((__half2*)Y)[i*4+2] = __hadd2(unpacked[0][1], adjust); // 45
  ((__half2*)Y)[i*4+1] = __hadd2(unpacked[1][0], adjust); // 23
  ((__half2*)Y)[i*4+3] = __hadd2(unpacked[1][1], adjust); // 67
#endif
}


void decompress_e8p_origorder(
    torch::Tensor YIs,      // m x (n/8)
    torch::Tensor CB,       // 256 x 8
    torch::Tensor &Y         // m x n
) {
#if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 800
  size_t m = Y.sizes()[0];
  size_t n = Y.sizes()[1];

  assert(YIs.is_contiguous());
  assert(CB.is_contiguous());
  assert(Y.is_contiguous());

  assert(YIs.sizes()[0] == m);
  assert(YIs.sizes()[1] * 8 == n);
  assert(CB.sizes()[0] == 256);

  const dim3 threads(DECOMPRESS_E8P_BLOCK_SIZE);
  const dim3 blocks(m*n/(8*DECOMPRESS_E8P_BLOCK_SIZE));
  hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
  cuda_decompress_e8p_origorder_kernel<<<blocks, threads, 0, stream>>>(
    YIs.data_ptr<int16_t>(),
    CB.data_ptr<int64_t>(),
    Y.data_ptr<c10::Half>()
  );
#endif
}

#define DECOMPRESS_HI_BLOCK_SIZE 256

__global__ void cuda_decompress_hi_origorder_kernel(
    const uint32_t* __restrict__ YIs,	  // m x (n/8)
    c10::Half* __restrict__ Y             // m x n
) {
#if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 800
  const long i = threadIdx.x + DECOMPRESS_HI_BLOCK_SIZE * blockIdx.x;
  uint32_t qa = YIs[i];

  const uint32_t c0 = 0x64086408;
  const half y16_ = __float2half_rn(1.0f / 16.0f);
  const half2 y16 = __halves2half2(y16_, y16_);
  const half z16_ = __float2half_rn(-1024.0f / 16.0f - 8.0f);
  const half2 z16 = __halves2half2(z16_, z16_);


  uint32_t q0 = (((qa & 0x000f000f) << 4) | c0);
  uint32_t q1 = ((qa & 0x00f000f0)| c0);
  qa >>= 8;
  uint32_t q2 = (((qa & 0x000f000f) << 4) | c0);
  uint32_t q3 = ((qa & 0x00f000f0) | c0);
  ((__half2*)Y)[i*4] = __hfma2(*((half2*)(&q0)), y16, z16);
  ((__half2*)Y)[i*4+1] = __hfma2(*((half2*)(&q1)), y16, z16);
  ((__half2*)Y)[i*4+2] = __hfma2(*((half2*)(&q2)), y16, z16);
  ((__half2*)Y)[i*4+3] = __hfma2(*((half2*)(&q3)), y16, z16);
#endif
}

void decompress_hi_origorder(
    torch::Tensor YIs,      // m x (n/8)
    torch::Tensor Y         // m x n
){
#if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 800
  size_t m = Y.sizes()[0];
  size_t n = Y.sizes()[1];

  assert(YIs.is_contiguous());
  assert(Y.is_contiguous());

  assert(YIs.sizes()[0] == m);
  assert(YIs.sizes()[1] * 8 == n);

  const dim3 threads(DECOMPRESS_HI_BLOCK_SIZE);
  const dim3 blocks(m*n/(8*DECOMPRESS_HI_BLOCK_SIZE));
  hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
  cuda_decompress_hi_origorder_kernel<<<blocks, threads, 0, stream>>>(
    (uint32_t*)YIs.data_ptr<int32_t>(),
    Y.data_ptr<c10::Half>()
  );
#endif
}
