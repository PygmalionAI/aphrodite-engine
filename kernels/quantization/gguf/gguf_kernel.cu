#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <torch/all.h>
#include <torch/python.h>
#include <c10/cuda/CUDAGuard.h>


#define QK_K 256
#define K_QUANTS_PER_ITERATION 2
#define WARP_SIZE 32
#define K_SCALE_SIZE 12
#define CUDA_DEQUANTIZE_BLOCK_SIZE 256
#define GGML_CUDA_DMMV_X 32
#define GGML_CUDA_MMV_Y 1

typedef half dfloat; // dequantize float
typedef half2 dfloat2;
typedef void (*dequantize_kernel_t)(const void * vx, const int ib, const int iqs, dfloat2 & v);
typedef void (*to_fp16_cuda_t)(const void * __restrict__ x, dfloat * __restrict__ y, int k, hipStream_t stream);

/**
 * Data Structures
 * QK = number of values after dequantization
 * QR = QK / number of values before dequantization
 * QI = number of 32 bit integers before dequantization
*/

#define QK4_0 32
#define QR4_0 2
#define QI4_0 (QK4_0 / (4 * QR4_0))
typedef struct {
    half    d;              // delta
    uint8_t qs[QK4_0 / 2];  // nibbles / quants
} block_q4_0;

#define QK4_1 32
#define QR4_1 2
#define QI4_1 (QK4_1 / (4 * QR4_1))
typedef struct {
    half2   dm;             // dm.x = delta, dm.y = min
    uint8_t qs[QK4_1 / 2];  // nibbles / quants
} block_q4_1;

#define QK5_0 32
#define QR5_0 2
#define QI5_0 (QK5_0 / (4 * QR5_0))
typedef struct {
    half d;                 // delta
    uint8_t qh[4];          // 5-th bit of quants
    uint8_t qs[QK5_0 / 2];  // nibbles / quants
} block_q5_0;

#define QK5_1 32
#define QR5_1 2
#define QI5_1 (QK5_1 / (4 * QR5_1))
typedef struct {
    half2 dm;               // dm.x = delta, dm.y = min
    uint8_t qh[4];          // 5-th bit of quants
    uint8_t qs[QK5_1 / 2];  // nibbles / quants
} block_q5_1;

#define QK8_0 32
#define QR8_0 1
#define QI8_0 (QK8_0 / (4 * QR8_0))
typedef struct {
    half    d;              // delta
    int8_t  qs[QK8_0];      // quants
} block_q8_0;

#define QK8_1 32
#define QR8_1 1
#define QI8_1 (QK8_1 / (4 * QR8_1))
typedef struct {
    half2   ds;             // ds.x = delta, ds.y = sum
    int8_t  qs[QK8_0];      // quants
} block_q8_1;

#define QR2_K 4
#define QI2_K (QK_K / (4*QR2_K))
typedef struct {
    uint8_t scales[QK_K/16]; // scales and mins, quantized with 4 bits
    uint8_t qs[QK_K/4];      // quants
    half2 dm;                // super-block scale for quantized scales/mins
} block_q2_K;

#define QR3_K 4
#define QI3_K (QK_K / (4*QR3_K))
typedef struct {
    uint8_t hmask[QK_K/8];     // quants - high bit
    uint8_t qs[QK_K/4];        // quants - low 2 bits
    uint8_t scales[K_SCALE_SIZE]; // scales, quantized with 6 bits
    half d;             // super-block scale
} block_q3_K;

#define QR4_K 2
#define QI4_K (QK_K / (4*QR4_K))
typedef struct {
    half2 dm;                  // super-block scale for quantized scales/mins
    uint8_t scales[3*QK_K/64]; // scales, quantized with 6 bits
    uint8_t qs[QK_K/2];        // 4--bit quants
} block_q4_K;

#define QR5_K 2
#define QI5_K (QK_K / (4*QR5_K))
typedef struct {
    half2 dm;                     // super-block scale for quantized scales/mins
    uint8_t scales[K_SCALE_SIZE]; // scales and mins, quantized with 6 bits
    uint8_t qh[QK_K/8];           // quants, high bit
    uint8_t qs[QK_K/2];           // quants, low 4 bits
} block_q5_K;

#define QR6_K 2
#define QI6_K (QK_K / (4*QR6_K))
typedef struct {
    uint8_t ql[QK_K/2];   // quants, lower 4 bits
    uint8_t qh[QK_K/4];   // quants, upper 2 bits
    int8_t  scales[QK_K/16]; // scales
    half    d;         // delta
} block_q6_K;

#define QR2_XXS 8
#define QI2_XXS (QK_K / (4*QR2_XXS))
typedef struct {
    half d;
    uint16_t qs[QK_K/8];
} block_iq2_xxs;

#define QR2_XS 8
#define QI2_XS (QK_K / (4*QR2_XS))
typedef struct {
    half d;
    uint16_t qs[QK_K/8];
    uint8_t  scales[QK_K/32];
} block_iq2_xs;

static const __device__ uint64_t iq2xxs_grid[256] = {
    0x0808080808080808, 0x080808080808082b, 0x0808080808081919, 0x0808080808082b08,
    0x0808080808082b2b, 0x0808080808190819, 0x0808080808191908, 0x08080808082b0808,
    0x08080808082b082b, 0x08080808082b2b08, 0x08080808082b2b2b, 0x0808080819080819,
    0x0808080819081908, 0x0808080819190808, 0x0808080819192b08, 0x08080808192b0819,
    0x08080808192b1908, 0x080808082b080808, 0x080808082b08082b, 0x080808082b082b2b,
    0x080808082b2b082b, 0x0808081908080819, 0x0808081908081908, 0x0808081908190808,
    0x0808081908191919, 0x0808081919080808, 0x080808192b081908, 0x080808192b192b08,
    0x0808082b08080808, 0x0808082b0808082b, 0x0808082b082b082b, 0x0808082b2b08082b,
    0x0808190808080819, 0x0808190808081908, 0x0808190808190808, 0x08081908082b0819,
    0x08081908082b1908, 0x0808190819080808, 0x080819081908082b, 0x0808190819082b08,
    0x08081908192b0808, 0x080819082b080819, 0x080819082b081908, 0x080819082b190808,
    0x080819082b2b1908, 0x0808191908080808, 0x080819190808082b, 0x0808191908082b08,
    0x08081919082b0808, 0x080819191908192b, 0x08081919192b2b19, 0x080819192b080808,
    0x080819192b190819, 0x0808192b08082b19, 0x0808192b08190808, 0x0808192b19080808,
    0x0808192b2b081908, 0x0808192b2b2b1908, 0x08082b0808080808, 0x08082b0808081919,
    0x08082b0808082b08, 0x08082b0808191908, 0x08082b08082b2b08, 0x08082b0819080819,
    0x08082b0819081908, 0x08082b0819190808, 0x08082b081919082b, 0x08082b082b082b08,
    0x08082b1908081908, 0x08082b1919080808, 0x08082b2b0808082b, 0x08082b2b08191908,
    0x0819080808080819, 0x0819080808081908, 0x0819080808190808, 0x08190808082b0819,
    0x0819080819080808, 0x08190808192b0808, 0x081908082b081908, 0x081908082b190808,
    0x081908082b191919, 0x0819081908080808, 0x0819081908082b08, 0x08190819082b0808,
    0x0819081919190808, 0x0819081919192b2b, 0x081908192b080808, 0x0819082b082b1908,
    0x0819082b19081919, 0x0819190808080808, 0x0819190808082b08, 0x08191908082b0808,
    0x08191908082b1919, 0x0819190819082b19, 0x081919082b080808, 0x0819191908192b08,
    0x08191919192b082b, 0x0819192b08080808, 0x0819192b0819192b, 0x08192b0808080819,
    0x08192b0808081908, 0x08192b0808190808, 0x08192b0819080808, 0x08192b082b080819,
    0x08192b1908080808, 0x08192b1908081919, 0x08192b192b2b0808, 0x08192b2b19190819,
    0x082b080808080808, 0x082b08080808082b, 0x082b080808082b2b, 0x082b080819081908,
    0x082b0808192b0819, 0x082b08082b080808, 0x082b08082b08082b, 0x082b0819082b2b19,
    0x082b081919082b08, 0x082b082b08080808, 0x082b082b0808082b, 0x082b190808080819,
    0x082b190808081908, 0x082b190808190808, 0x082b190819080808, 0x082b19081919192b,
    0x082b191908080808, 0x082b191919080819, 0x082b1919192b1908, 0x082b192b2b190808,
    0x082b2b0808082b08, 0x082b2b08082b0808, 0x082b2b082b191908, 0x082b2b2b19081908,
    0x1908080808080819, 0x1908080808081908, 0x1908080808190808, 0x1908080808192b08,
    0x19080808082b0819, 0x19080808082b1908, 0x1908080819080808, 0x1908080819082b08,
    0x190808081919192b, 0x19080808192b0808, 0x190808082b080819, 0x190808082b081908,
    0x190808082b190808, 0x1908081908080808, 0x19080819082b0808, 0x19080819192b0819,
    0x190808192b080808, 0x190808192b081919, 0x1908082b08080819, 0x1908082b08190808,
    0x1908082b19082b08, 0x1908082b1919192b, 0x1908082b192b2b08, 0x1908190808080808,
    0x1908190808082b08, 0x19081908082b0808, 0x190819082b080808, 0x190819082b192b19,
    0x190819190819082b, 0x19081919082b1908, 0x1908192b08080808, 0x19082b0808080819,
    0x19082b0808081908, 0x19082b0808190808, 0x19082b0819080808, 0x19082b0819081919,
    0x19082b1908080808, 0x19082b1919192b08, 0x19082b19192b0819, 0x19082b192b08082b,
    0x19082b2b19081919, 0x19082b2b2b190808, 0x1919080808080808, 0x1919080808082b08,
    0x1919080808190819, 0x1919080808192b19, 0x19190808082b0808, 0x191908082b080808,
    0x191908082b082b08, 0x1919081908081908, 0x191908191908082b, 0x191908192b2b1908,
    0x1919082b2b190819, 0x191919082b190808, 0x191919082b19082b, 0x1919191908082b2b,
    0x1919192b08080819, 0x1919192b19191908, 0x19192b0808080808, 0x19192b0808190819,
    0x19192b0808192b19, 0x19192b08192b1908, 0x19192b1919080808, 0x19192b2b08082b08,
    0x192b080808081908, 0x192b080808190808, 0x192b080819080808, 0x192b0808192b2b08,
    0x192b081908080808, 0x192b081919191919, 0x192b082b08192b08, 0x192b082b192b0808,
    0x192b190808080808, 0x192b190808081919, 0x192b191908190808, 0x192b19190819082b,
    0x192b19192b081908, 0x192b2b081908082b, 0x2b08080808080808, 0x2b0808080808082b,
    0x2b08080808082b2b, 0x2b08080819080819, 0x2b0808082b08082b, 0x2b08081908081908,
    0x2b08081908192b08, 0x2b08081919080808, 0x2b08082b08190819, 0x2b08190808080819,
    0x2b08190808081908, 0x2b08190808190808, 0x2b08190808191919, 0x2b08190819080808,
    0x2b081908192b0808, 0x2b08191908080808, 0x2b0819191908192b, 0x2b0819192b191908,
    0x2b08192b08082b19, 0x2b08192b19080808, 0x2b08192b192b0808, 0x2b082b080808082b,
    0x2b082b1908081908, 0x2b082b2b08190819, 0x2b19080808081908, 0x2b19080808190808,
    0x2b190808082b1908, 0x2b19080819080808, 0x2b1908082b2b0819, 0x2b1908190819192b,
    0x2b1908192b080808, 0x2b19082b19081919, 0x2b19190808080808, 0x2b191908082b082b,
    0x2b19190819081908, 0x2b19191919190819, 0x2b192b082b080819, 0x2b192b19082b0808,
    0x2b2b08080808082b, 0x2b2b080819190808, 0x2b2b08082b081919, 0x2b2b081908082b19,
    0x2b2b082b08080808, 0x2b2b190808192b08, 0x2b2b2b0819190808, 0x2b2b2b1908081908,
};

static const __device__ uint64_t iq2xs_grid[512] = {
    0x0808080808080808, 0x080808080808082b, 0x0808080808081919, 0x0808080808082b08,
    0x0808080808082b2b, 0x0808080808190819, 0x0808080808191908, 0x080808080819192b,
    0x0808080808192b19, 0x08080808082b0808, 0x08080808082b082b, 0x08080808082b1919,
    0x08080808082b2b08, 0x0808080819080819, 0x0808080819081908, 0x080808081908192b,
    0x0808080819082b19, 0x0808080819190808, 0x080808081919082b, 0x0808080819191919,
    0x0808080819192b08, 0x08080808192b0819, 0x08080808192b1908, 0x080808082b080808,
    0x080808082b08082b, 0x080808082b081919, 0x080808082b082b08, 0x080808082b190819,
    0x080808082b191908, 0x080808082b192b19, 0x080808082b2b0808, 0x0808081908080819,
    0x0808081908081908, 0x080808190808192b, 0x0808081908082b19, 0x0808081908190808,
    0x080808190819082b, 0x0808081908191919, 0x0808081908192b08, 0x0808081908192b2b,
    0x08080819082b0819, 0x08080819082b1908, 0x0808081919080808, 0x080808191908082b,
    0x0808081919081919, 0x0808081919082b08, 0x0808081919190819, 0x0808081919191908,
    0x08080819192b0808, 0x08080819192b2b08, 0x080808192b080819, 0x080808192b081908,
    0x080808192b190808, 0x0808082b08080808, 0x0808082b0808082b, 0x0808082b08081919,
    0x0808082b08082b08, 0x0808082b08190819, 0x0808082b08191908, 0x0808082b082b0808,
    0x0808082b19080819, 0x0808082b19081908, 0x0808082b19190808, 0x0808082b19191919,
    0x0808082b2b080808, 0x0808082b2b082b2b, 0x0808190808080819, 0x0808190808081908,
    0x080819080808192b, 0x0808190808082b19, 0x0808190808190808, 0x080819080819082b,
    0x0808190808191919, 0x0808190808192b08, 0x08081908082b0819, 0x08081908082b1908,
    0x0808190819080808, 0x080819081908082b, 0x0808190819081919, 0x0808190819082b08,
    0x0808190819190819, 0x0808190819191908, 0x080819081919192b, 0x08081908192b0808,
    0x080819082b080819, 0x080819082b081908, 0x080819082b190808, 0x0808191908080808,
    0x080819190808082b, 0x0808191908081919, 0x0808191908082b08, 0x0808191908190819,
    0x0808191908191908, 0x08081919082b0808, 0x0808191919080819, 0x0808191919081908,
    0x0808191919190808, 0x08081919192b0819, 0x080819192b080808, 0x0808192b08080819,
    0x0808192b08081908, 0x0808192b08190808, 0x0808192b082b192b, 0x0808192b19080808,
    0x0808192b1908082b, 0x0808192b2b081908, 0x08082b0808080808, 0x08082b080808082b,
    0x08082b0808081919, 0x08082b0808082b08, 0x08082b0808082b2b, 0x08082b0808190819,
    0x08082b0808191908, 0x08082b08082b0808, 0x08082b08082b1919, 0x08082b0819080819,
    0x08082b0819081908, 0x08082b0819190808, 0x08082b0819192b08, 0x08082b082b080808,
    0x08082b082b2b0808, 0x08082b082b2b2b2b, 0x08082b1908080819, 0x08082b1908081908,
    0x08082b1908190808, 0x08082b1919080808, 0x08082b192b080819, 0x08082b192b082b19,
    0x08082b2b08080808, 0x08082b2b082b0808, 0x08082b2b082b2b08, 0x08082b2b2b19192b,
    0x08082b2b2b2b0808, 0x0819080808080819, 0x0819080808081908, 0x081908080808192b,
    0x0819080808082b19, 0x0819080808190808, 0x081908080819082b, 0x0819080808191919,
    0x0819080808192b08, 0x08190808082b0819, 0x08190808082b1908, 0x0819080819080808,
    0x081908081908082b, 0x0819080819081919, 0x0819080819082b08, 0x0819080819190819,
    0x0819080819191908, 0x08190808192b0808, 0x08190808192b2b2b, 0x081908082b080819,
    0x081908082b081908, 0x081908082b190808, 0x0819081908080808, 0x081908190808082b,
    0x0819081908081919, 0x0819081908082b08, 0x0819081908190819, 0x0819081908191908,
    0x08190819082b0808, 0x0819081919080819, 0x0819081919081908, 0x0819081919190808,
    0x081908192b080808, 0x081908192b191908, 0x081908192b19192b, 0x0819082b08080819,
    0x0819082b08081908, 0x0819082b0808192b, 0x0819082b08190808, 0x0819082b19080808,
    0x0819082b192b0808, 0x0819190808080808, 0x081919080808082b, 0x0819190808081919,
    0x0819190808082b08, 0x0819190808190819, 0x0819190808191908, 0x08191908082b0808,
    0x0819190819080819, 0x0819190819081908, 0x0819190819082b19, 0x0819190819190808,
    0x08191908192b1908, 0x081919082b080808, 0x0819191908080819, 0x0819191908081908,
    0x0819191908190808, 0x0819191919080808, 0x0819192b08080808, 0x0819192b08191908,
    0x0819192b19082b19, 0x08192b0808080819, 0x08192b0808081908, 0x08192b0808190808,
    0x08192b080819082b, 0x08192b0819080808, 0x08192b0819191908, 0x08192b082b08192b,
    0x08192b1908080808, 0x08192b1908081919, 0x08192b19192b192b, 0x08192b2b19190819,
    0x08192b2b2b2b2b19, 0x082b080808080808, 0x082b08080808082b, 0x082b080808081919,
    0x082b080808082b08, 0x082b080808082b2b, 0x082b080808190819, 0x082b080808191908,
    0x082b0808082b0808, 0x082b080819080819, 0x082b080819081908, 0x082b080819190808,
    0x082b08082b080808, 0x082b08082b2b0808, 0x082b081908080819, 0x082b081908081908,
    0x082b081908190808, 0x082b081919080808, 0x082b081919082b08, 0x082b0819192b1919,
    0x082b082b08080808, 0x082b082b082b082b, 0x082b082b2b080808, 0x082b082b2b2b2b08,
    0x082b190808080819, 0x082b190808081908, 0x082b190808190808, 0x082b1908082b2b19,
    0x082b190819080808, 0x082b191908080808, 0x082b191919080819, 0x082b19191919082b,
    0x082b19192b192b19, 0x082b192b08080819, 0x082b192b08192b2b, 0x082b192b2b2b192b,
    0x082b2b0808080808, 0x082b2b0808082b08, 0x082b2b0808082b2b, 0x082b2b08082b0808,
    0x082b2b0819191919, 0x082b2b082b082b08, 0x082b2b082b2b082b, 0x082b2b19192b2b08,
    0x082b2b192b190808, 0x082b2b2b08082b08, 0x082b2b2b082b0808, 0x082b2b2b2b08082b,
    0x082b2b2b2b082b08, 0x082b2b2b2b082b2b, 0x1908080808080819, 0x1908080808081908,
    0x190808080808192b, 0x1908080808082b19, 0x1908080808190808, 0x190808080819082b,
    0x1908080808191919, 0x1908080808192b08, 0x19080808082b0819, 0x19080808082b1908,
    0x1908080819080808, 0x190808081908082b, 0x1908080819081919, 0x1908080819082b08,
    0x1908080819082b2b, 0x1908080819190819, 0x1908080819191908, 0x19080808192b0808,
    0x19080808192b1919, 0x190808082b080819, 0x190808082b081908, 0x190808082b190808,
    0x1908081908080808, 0x190808190808082b, 0x1908081908081919, 0x1908081908082b08,
    0x1908081908190819, 0x1908081908191908, 0x19080819082b0808, 0x1908081919080819,
    0x1908081919081908, 0x1908081919190808, 0x190808192b080808, 0x190808192b081919,
    0x190808192b2b082b, 0x1908082b08080819, 0x1908082b08081908, 0x1908082b08190808,
    0x1908082b0819082b, 0x1908082b082b2b19, 0x1908082b19080808, 0x1908190808080808,
    0x190819080808082b, 0x1908190808081919, 0x1908190808082b08, 0x1908190808190819,
    0x1908190808191908, 0x1908190808192b19, 0x19081908082b0808, 0x1908190819080819,
    0x1908190819081908, 0x1908190819190808, 0x190819082b080808, 0x190819082b191908,
    0x1908191908080819, 0x1908191908081908, 0x1908191908190808, 0x19081919082b1908,
    0x1908191919080808, 0x190819192b192b2b, 0x1908192b08080808, 0x1908192b08082b2b,
    0x1908192b19081908, 0x1908192b19190808, 0x19082b0808080819, 0x19082b0808081908,
    0x19082b0808190808, 0x19082b0819080808, 0x19082b0819081919, 0x19082b0819191908,
    0x19082b08192b082b, 0x19082b1908080808, 0x19082b1908190819, 0x19082b1919081908,
    0x19082b1919190808, 0x19082b19192b2b19, 0x19082b2b08081908, 0x1919080808080808,
    0x191908080808082b, 0x1919080808081919, 0x1919080808082b08, 0x1919080808190819,
    0x1919080808191908, 0x19190808082b0808, 0x19190808082b2b08, 0x1919080819080819,
    0x1919080819081908, 0x1919080819190808, 0x191908082b080808, 0x1919081908080819,
    0x1919081908081908, 0x1919081908190808, 0x1919081908191919, 0x1919081919080808,
    0x191908191908082b, 0x1919082b08080808, 0x1919082b19081908, 0x1919082b2b2b2b2b,
    0x1919190808080819, 0x1919190808081908, 0x1919190808190808, 0x19191908082b0819,
    0x1919190819080808, 0x19191908192b0808, 0x191919082b080819, 0x191919082b2b0819,
    0x1919191908080808, 0x1919191908082b08, 0x191919192b080808, 0x191919192b082b08,
    0x1919192b082b0819, 0x1919192b192b2b08, 0x1919192b2b2b0819, 0x19192b0808080808,
    0x19192b0808191908, 0x19192b0819080819, 0x19192b0819190808, 0x19192b082b192b19,
    0x19192b1908192b2b, 0x19192b1919080808, 0x19192b191908082b, 0x19192b2b2b081919,
    0x192b080808080819, 0x192b080808081908, 0x192b080808190808, 0x192b080819080808,
    0x192b080819191908, 0x192b0808192b082b, 0x192b08082b08192b, 0x192b08082b2b2b19,
    0x192b081908080808, 0x192b082b082b1908, 0x192b082b19082b2b, 0x192b082b2b19082b,
    0x192b190808080808, 0x192b19080819192b, 0x192b191908190808, 0x192b191919080808,
    0x192b191919081919, 0x192b19192b2b1908, 0x192b2b0808080819, 0x192b2b08192b2b2b,
    0x192b2b19082b1919, 0x192b2b2b0808192b, 0x192b2b2b19191908, 0x192b2b2b192b082b,
    0x2b08080808080808, 0x2b0808080808082b, 0x2b08080808081919, 0x2b08080808082b08,
    0x2b08080808190819, 0x2b08080808191908, 0x2b080808082b0808, 0x2b080808082b2b2b,
    0x2b08080819080819, 0x2b08080819081908, 0x2b08080819190808, 0x2b0808082b080808,
    0x2b0808082b08082b, 0x2b0808082b2b2b08, 0x2b0808082b2b2b2b, 0x2b08081908080819,
    0x2b08081908081908, 0x2b0808190808192b, 0x2b08081908190808, 0x2b08081919080808,
    0x2b08081919190819, 0x2b08081919192b19, 0x2b08082b08080808, 0x2b08082b082b0808,
    0x2b08082b2b080808, 0x2b08082b2b08082b, 0x2b08082b2b2b0808, 0x2b08082b2b2b2b08,
    0x2b08190808080819, 0x2b08190808081908, 0x2b08190808190808, 0x2b0819080819082b,
    0x2b08190808191919, 0x2b08190819080808, 0x2b081908192b0808, 0x2b0819082b082b19,
    0x2b08191908080808, 0x2b08191919081908, 0x2b0819192b2b1919, 0x2b08192b08192b08,
    0x2b08192b192b2b2b, 0x2b082b0808080808, 0x2b082b0808082b08, 0x2b082b08082b1919,
    0x2b082b0819192b2b, 0x2b082b082b080808, 0x2b082b082b08082b, 0x2b082b082b2b2b08,
    0x2b082b190808192b, 0x2b082b2b082b082b, 0x2b082b2b2b080808, 0x2b082b2b2b082b08,
    0x2b082b2b2b19192b, 0x2b082b2b2b2b2b08, 0x2b19080808080819, 0x2b19080808081908,
    0x2b19080808190808, 0x2b19080819080808, 0x2b1908081919192b, 0x2b1908082b081908,
    0x2b19081908080808, 0x2b190819082b082b, 0x2b190819192b1908, 0x2b19082b1919192b,
    0x2b19082b2b082b19, 0x2b19190808080808, 0x2b19190808081919, 0x2b19190819081908,
    0x2b19190819190808, 0x2b19190819192b08, 0x2b191919082b2b19, 0x2b1919192b190808,
    0x2b1919192b19082b, 0x2b19192b19080819, 0x2b192b0819190819, 0x2b192b082b2b192b,
    0x2b192b1919082b19, 0x2b192b2b08191919, 0x2b192b2b192b0808, 0x2b2b080808080808,
    0x2b2b08080808082b, 0x2b2b080808082b08, 0x2b2b080808082b2b, 0x2b2b0808082b0808,
    0x2b2b0808082b2b2b, 0x2b2b08082b2b0808, 0x2b2b081919190819, 0x2b2b081919192b19,
    0x2b2b08192b2b192b, 0x2b2b082b08080808, 0x2b2b082b0808082b, 0x2b2b082b08082b08,
    0x2b2b082b082b2b2b, 0x2b2b082b2b080808, 0x2b2b082b2b2b0808, 0x2b2b190819080808,
    0x2b2b19082b191919, 0x2b2b192b192b1919, 0x2b2b192b2b192b08, 0x2b2b2b0808082b2b,
    0x2b2b2b08082b0808, 0x2b2b2b08082b082b, 0x2b2b2b08082b2b08, 0x2b2b2b082b2b0808,
    0x2b2b2b082b2b2b08, 0x2b2b2b1908081908, 0x2b2b2b192b081908, 0x2b2b2b192b08192b,
    0x2b2b2b2b082b2b08, 0x2b2b2b2b082b2b2b, 0x2b2b2b2b2b190819, 0x2b2b2b2b2b2b2b2b,
};

static const __device__ uint8_t ksigns_iq2xs[128] = {
      0, 129, 130,   3, 132,   5,   6, 135, 136,   9,  10, 139,  12, 141, 142,  15,
    144,  17,  18, 147,  20, 149, 150,  23,  24, 153, 154,  27, 156,  29,  30, 159,
    160,  33,  34, 163,  36, 165, 166,  39,  40, 169, 170,  43, 172,  45,  46, 175,
     48, 177, 178,  51, 180,  53,  54, 183, 184,  57,  58, 187,  60, 189, 190,  63,
    192,  65,  66, 195,  68, 197, 198,  71,  72, 201, 202,  75, 204,  77,  78, 207,
     80, 209, 210,  83, 212,  85,  86, 215, 216,  89,  90, 219,  92, 221, 222,  95,
     96, 225, 226,  99, 228, 101, 102, 231, 232, 105, 106, 235, 108, 237, 238, 111,
    240, 113, 114, 243, 116, 245, 246, 119, 120, 249, 250, 123, 252, 125, 126, 255,
};

static const __device__ uint8_t kmask_iq2xs[8] = {1, 2, 4, 8, 16, 32, 64, 128};


// Dequant functions
static __device__ __forceinline__ void dequantize_q4_0(const void * vx, const int ib, const int iqs, dfloat2 & v){
    const block_q4_0 * x = (const block_q4_0 *) vx;

    const dfloat d = x[ib].d;

    const int vui = x[ib].qs[iqs];

    v.x = __int2half_rn(vui & 0xF);
    v.y = __int2half_rn(vui >> 4);

    v = __hsub2(v, __floats2half2_rn(8.0f, 8.0f));
    v = __hmul2(v, {d, d});
}

static __device__ __forceinline__ void dequantize_q4_1(const void * vx, const int ib, const int iqs, dfloat2 & v){
    const block_q4_1 * x = (const block_q4_1 *) vx;

    const dfloat d = __low2half(x[ib].dm);
    const dfloat m = __high2half(x[ib].dm);

    const int vui = x[ib].qs[iqs];

    v.x = __int2half_rn(vui & 0xF);
    v.y = __int2half_rn(vui >> 4);

    v = __hmul2(v, {d, d});
    v = __hadd2(v, {m, m});
}

static __device__ __forceinline__ void dequantize_q5_0(const void * vx, const int ib, const int iqs, dfloat2 & v){
    const block_q5_0 * x = (const block_q5_0 *) vx;

    const dfloat d = x[ib].d;

    uint32_t qh;
    memcpy(&qh, x[ib].qh, sizeof(qh));

    const int xh_0 = ((qh >> (iqs +  0)) << 4) & 0x10;
    const int xh_1 = ((qh >> (iqs + 12))     ) & 0x10;

    v.x = __int2half_rn((x[ib].qs[iqs] & 0xf) | xh_0);
    v.y = __int2half_rn((x[ib].qs[iqs] >>  4) | xh_1);

    v = __hsub2(v, __floats2half2_rn(16.0f, 16.0f));
    v = __hmul2(v, {d, d});
}

static __device__ __forceinline__ void dequantize_q5_1(const void * vx, const int ib, const int iqs, dfloat2 & v){
    const block_q5_1 * x = (const block_q5_1 *) vx;

    const dfloat d = __low2half(x[ib].dm);
    const dfloat m = __high2half(x[ib].dm);

    uint32_t qh;
    memcpy(&qh, x[ib].qh, sizeof(qh));

    const int xh_0 = ((qh >> (iqs +  0)) << 4) & 0x10;
    const int xh_1 = ((qh >> (iqs + 12))     ) & 0x10;

    v.x = __int2half_rn((x[ib].qs[iqs] & 0xf) | xh_0);
    v.y = __int2half_rn((x[ib].qs[iqs] >>  4) | xh_1);

    v = __hmul2(v, {d, d});
    v = __hadd2(v, {m, m});
}

static __device__ __forceinline__ void dequantize_q8_0(const void * vx, const int ib, const int iqs, dfloat2 & v){
    const block_q8_0 * x = (const block_q8_0 *) vx;

    const dfloat d = x[ib].d;

    v.x = __int2half_rn(x[ib].qs[iqs + 0]);
    v.y = __int2half_rn(x[ib].qs[iqs + 1]);

    v = __hmul2(v, {d, d});
}

template <int qk, int qr, dequantize_kernel_t dequantize_kernel, typename dst_t>
static __global__ void dequantize_block(const void * __restrict__ vx, dst_t * __restrict__ y, const int k) {
    const int i = 2*(blockDim.x*blockIdx.x + threadIdx.x);

    if (i >= k) {
        return;
    }

    const int ib = i/qk; // block index
    const int iqs = (i%qk)/qr; // quant index
    const int iybs = i - i%qk; // y block start index
    const int y_offset = qr == 1 ? 1 : qk/2;

    // dequantize
    dfloat2 v;
    dequantize_kernel(vx, ib, iqs, v);

    y[iybs + iqs + 0]        = v.x;
    y[iybs + iqs + y_offset] = v.y;
}

template<typename dst_t>
static __global__ void dequantize_block_q2_K(const void * __restrict__ vx, dst_t * __restrict__ yy) {

    const int i   = blockIdx.x;
    const block_q2_K * x = (const block_q2_K *) vx;

    const int tid = threadIdx.x;
    const int n   = tid/32;
    const int l   = tid - 32*n;
    const int is  = 8*n + l/16;

    const uint8_t q = x[i].qs[32*n + l];
    dst_t * y = yy + i*QK_K + 128*n;

    half dall = __low2half(x[i].dm);
    half dmin = __high2half(x[i].dm);
    y[l+ 0] = __hsub(__hmul(dall, __int2half_rn((x[i].scales[is+0] & 0xF) * ((q >> 0) & 3))), __hmul(dmin,  __int2half_rn(x[i].scales[is+0] >> 4)));
    y[l+32] = __hsub(__hmul(dall, __int2half_rn((x[i].scales[is+2] & 0xF) * ((q >> 2) & 3))), __hmul(dmin,  __int2half_rn(x[i].scales[is+2] >> 4)));
    y[l+64] = __hsub(__hmul(dall, __int2half_rn((x[i].scales[is+4] & 0xF) * ((q >> 4) & 3))), __hmul(dmin,  __int2half_rn(x[i].scales[is+4] >> 4)));
    y[l+96] = __hsub(__hmul(dall, __int2half_rn((x[i].scales[is+6] & 0xF) * ((q >> 6) & 3))), __hmul(dmin,  __int2half_rn(x[i].scales[is+6] >> 4)));
}

template<typename dst_t>
static __global__ void dequantize_block_q3_K(const void * __restrict__ vx, dst_t * __restrict__ yy) {

    const int i = blockIdx.x;
    const block_q3_K * x = (const block_q3_K *) vx;

    const int r = threadIdx.x/4;
    const int tid = r/2;
    const int is0 = r%2;
    const int l0 = 16*is0 + 4*(threadIdx.x%4);
    const int n = tid / 4;
    const int j = tid - 4*n;

    uint8_t m = 1 << (4*n + j);
    int is = 8*n + 2*j + is0;
    int shift = 2*j;

    int8_t us = is <  4 ? (x[i].scales[is-0] & 0xF) | (((x[i].scales[is+8] >> 0) & 3) << 4) :
                is <  8 ? (x[i].scales[is-0] & 0xF) | (((x[i].scales[is+4] >> 2) & 3) << 4) :
                is < 12 ? (x[i].scales[is-8] >>  4) | (((x[i].scales[is+0] >> 4) & 3) << 4) :
                          (x[i].scales[is-8] >>  4) | (((x[i].scales[is-4] >> 6) & 3) << 4);
    half d_all = x[i].d;
    half dl = __hmul(d_all,  __int2half_rn(us - 32));

    dst_t * y = yy + i*QK_K + 128*n + 32*j;
    const uint8_t * q = x[i].qs + 32*n;
    const uint8_t * hm = x[i].hmask;

    for (int l = l0; l < l0+4; ++l) y[l] = __hmul(dl,  __int2half_rn((int8_t)((q[l] >> shift) & 3) - ((hm[l] & m) ? 0 : 4)));
}

static inline __device__ void get_scale_min_k4(int j, const uint8_t * q, uint8_t & d, uint8_t & m) {
    if (j < 4) {
        d = q[j] & 63; m = q[j + 4] & 63;
    } else {
        d = (q[j+4] & 0xF) | ((q[j-4] >> 6) << 4);
        m = (q[j+4] >>  4) | ((q[j-0] >> 6) << 4);
    }
}

template<typename dst_t>
static __global__ void dequantize_block_q4_K(const void * __restrict__ vx, dst_t * __restrict__ yy) {
    const block_q4_K * x = (const block_q4_K *) vx;

    const int i = blockIdx.x;

    // assume 32 threads
    const int tid = threadIdx.x;
    const int il  = tid/8;
    const int ir  = tid%8;
    const int is  = 2*il;
    const int n   = 4;

    dst_t * y = yy + i*QK_K + 64*il + n*ir;

    const half dall = __low2half(x[i].dm);
    const half dmin = __high2half(x[i].dm);

    const uint8_t * q = x[i].qs + 32*il + n*ir;

    uint8_t sc, m;
    get_scale_min_k4(is + 0, x[i].scales, sc, m);
    const half d1 = __hmul(dall, __int2half_rn(sc));
    const half m1 = __hmul(dmin,  __int2half_rn(m));
    get_scale_min_k4(is + 1, x[i].scales, sc, m);
    const half d2 = __hmul(dall, __int2half_rn(sc));
    const half m2 = __hmul(dmin, __int2half_rn(m));
    for (int l = 0; l < n; ++l) {
        y[l + 0] = __hsub(__hmul(d1, __int2half_rn(q[l] & 0xF)), m1);
        y[l +32] = __hsub(__hmul(d2,  __int2half_rn(q[l] >> 4)), m2);
    }
}

template<typename dst_t>
static __global__ void dequantize_block_q5_K(const void * __restrict__ vx, dst_t * __restrict__ yy) {
    const block_q5_K * x = (const block_q5_K *) vx;

    const int i = blockIdx.x;

    // assume 64 threads - this is very slightly better than the one below
    const int tid = threadIdx.x;
    const int il  = tid/16;   // il is in 0...3
    const int ir  = tid%16;   // ir is in 0...15
    const int is  = 2*il;     // is is in 0...6

    dst_t * y = yy + i*QK_K + 64*il + 2*ir;

    const half dall = __low2half(x[i].dm);
    const half dmin = __high2half(x[i].dm);

    const uint8_t * ql = x[i].qs + 32*il + 2*ir;
    const uint8_t * qh = x[i].qh + 2*ir;

    uint8_t sc, m;
    get_scale_min_k4(is + 0, x[i].scales, sc, m);
    const half d1 = __hmul(dall, __int2half_rn(sc)); const half m1 = __hmul(dmin, __int2half_rn(m));
    get_scale_min_k4(is + 1, x[i].scales, sc, m);
    const half d2 = __hmul(dall, __int2half_rn(sc)); const half m2 = __hmul(dmin, __int2half_rn(m));

    uint8_t   hm  = 1 << (2*il);
    y[ 0] = __hsub(__hmul(d1, __int2half_rn((ql[0] & 0xF) + (qh[0] & hm ? 16 : 0))), m1);
    y[ 1] = __hsub(__hmul(d1, __int2half_rn((ql[1] & 0xF) + (qh[1] & hm ? 16 : 0))), m1);
    hm <<= 1;
    y[32] = __hsub(__hmul(d2, __int2half_rn((ql[0] >>  4) + (qh[0] & hm ? 16 : 0))), m2);
    y[33] = __hsub(__hmul(d2, __int2half_rn((ql[1] >>  4) + (qh[1] & hm ? 16 : 0))), m2);
}

template<typename dst_t>
static __global__ void dequantize_block_q6_K(const void * __restrict__ vx, dst_t * __restrict__ yy) {
    const block_q6_K * x = (const block_q6_K *) vx;

    const int i = blockIdx.x;

    // assume 64 threads - this is very slightly better than the one below
    const int tid = threadIdx.x;
    const int ip  = tid/32;   // ip is 0 or 1
    const int il  = tid - 32*ip; // 0...32
    const int is  = 8*ip + il/16;

    dst_t * y = yy + i*QK_K + 128*ip + il;

    const half d = x[i].d;

    const uint8_t * ql = x[i].ql + 64*ip + il;
    const uint8_t   qh = x[i].qh[32*ip + il];
    const int8_t  * sc = x[i].scales + is;

    y[ 0] = __hmul(d, __int2half_rn(sc[0] * ((int8_t)((ql[ 0] & 0xF) | (((qh >> 0) & 3) << 4)) - 32)));
    y[32] = __hmul(d, __int2half_rn(sc[2] * ((int8_t)((ql[32] & 0xF) | (((qh >> 2) & 3) << 4)) - 32)));
    y[64] = __hmul(d, __int2half_rn(sc[4] * ((int8_t)((ql[ 0]  >> 4) | (((qh >> 4) & 3) << 4)) - 32)));
    y[96] = __hmul(d, __int2half_rn(sc[6] * ((int8_t)((ql[32]  >> 4) | (((qh >> 6) & 3) << 4)) - 32)));
}

template<typename dst_t>
static __global__ void dequantize_block_iq2_xxs(const void * __restrict__ vx, dst_t * __restrict__ yy) {

    const int i   = blockIdx.x;
    const block_iq2_xxs * x = (const block_iq2_xxs  *) vx;

    const int tid = threadIdx.x;
    const int il = tid/8; // 0...3
    const int ib = tid%8; // 0...7
    dst_t * y = yy + i*QK_K + 32*ib + 8*il;
    const uint16_t * q2 = x[i].qs + 4*ib;
    const uint8_t  * aux8 = (const uint8_t *)q2;
    const uint8_t  * grid = (const uint8_t *)(iq2xxs_grid + aux8[il]);
    const uint32_t aux32 = q2[2] | (q2[3] << 16);
    const float d = __half2float(x[i].d) * (0.5f + (aux32 >> 28)) * 0.25f;
    const uint8_t signs = ksigns_iq2xs[(aux32 >> 7*il) & 127];
    for (int j = 0; j < 8; ++j) y[j] = __float2half(d * grid[j] * (signs & kmask_iq2xs[j] ? -1.f : 1.f));
}

template<typename dst_t>
static __global__ void dequantize_block_iq2_xs(const void * __restrict__ vx, dst_t * __restrict__ yy) {

    const int i   = blockIdx.x;
    const block_iq2_xs * x = (const block_iq2_xs *) vx;

    const int tid = threadIdx.x;
    const int il = tid/8; // 0...3
    const int ib = tid%8; // 0...7
    dst_t * y = yy + i*QK_K + 32*ib + 8*il;
    const uint16_t * q2 = x[i].qs + 4*ib;
    const uint8_t  * grid = (const uint8_t *)(iq2xs_grid + (q2[il] & 511));
    const float d = __half2float(x[i].d) * (0.5f + ((x[i].scales[ib] >> 4*(il/2)) & 0xf)) * 0.25f;
    const uint8_t signs = ksigns_iq2xs[q2[il] >> 9];
    for (int j = 0; j < 8; ++j) y[j] = __float2half(d * grid[j] * (signs & kmask_iq2xs[j] ? -1.f : 1.f));

}

template <int qk, int qr, dequantize_kernel_t dequantize_kernel, typename dst_t>
static void dequantize_block_cuda(const void * __restrict__ vx, dst_t * __restrict__ y, const int k, hipStream_t stream) {
    const int num_blocks = (k + 2*CUDA_DEQUANTIZE_BLOCK_SIZE - 1) / (2*CUDA_DEQUANTIZE_BLOCK_SIZE);
    dequantize_block<qk, qr, dequantize_kernel><<<num_blocks, CUDA_DEQUANTIZE_BLOCK_SIZE, 0, stream>>>(vx, y, k);
}

template<typename dst_t>
static void dequantize_row_q2_K_cuda(const void * vx, dst_t * y, const int k, hipStream_t stream) {
    const int nb = k / QK_K;
    dequantize_block_q2_K<<<nb, 64, 0, stream>>>(vx, y);
}

template<typename dst_t>
static void dequantize_row_q3_K_cuda(const void * vx, dst_t * y, const int k, hipStream_t stream) {
    const int nb = k / QK_K;
    dequantize_block_q3_K<<<nb, 64, 0, stream>>>(vx, y);
}

template<typename dst_t>
static void dequantize_row_q4_K_cuda(const void * vx, dst_t * y, const int k, hipStream_t stream) {
    const int nb = k / QK_K;
    dequantize_block_q4_K<<<nb, 32, 0, stream>>>(vx, y);
}

template<typename dst_t>
static void dequantize_row_q5_K_cuda(const void * vx, dst_t * y, const int k, hipStream_t stream) {
    const int nb = k / QK_K;
    dequantize_block_q5_K<<<nb, 64, 0, stream>>>(vx, y);
}

template<typename dst_t>
static void dequantize_row_q6_K_cuda(const void * vx, dst_t * y, const int k, hipStream_t stream) {
    const int nb = k / QK_K;
    dequantize_block_q6_K<<<nb, 64, 0, stream>>>(vx, y);
}

template<typename dst_t>
static void dequantize_row_iq2_xxs_cuda(const void * vx, dst_t * y, const int k, hipStream_t stream) {
    const int nb = k / QK_K;
    dequantize_block_iq2_xxs<<<nb, 32, 0, stream>>>(vx, y);
}

template<typename dst_t>
static void dequantize_row_iq2_xs_cuda(const void * vx, dst_t * y, const int k, hipStream_t stream) {
    const int nb = k / QK_K;
    dequantize_block_iq2_xs<<<nb, 32, 0, stream>>>(vx, y);
}

static to_fp16_cuda_t ggml_get_to_fp16_cuda(int type) {
    switch (type) {
        case 2:
            return dequantize_block_cuda<QK4_0, QR4_0, dequantize_q4_0>;
        case 3:
            return dequantize_block_cuda<QK4_1, QR4_1, dequantize_q4_1>;
        case 6:
            return dequantize_block_cuda<QK5_0, QR5_0, dequantize_q5_0>;
        case 7:
            return dequantize_block_cuda<QK5_1, QR5_1, dequantize_q5_1>;
        case 8:
            return dequantize_block_cuda<QK8_0, QR8_0, dequantize_q8_0>;
        case 10:
            return dequantize_row_q2_K_cuda;
        case 11:
            return dequantize_row_q3_K_cuda;
        case 12:
            return dequantize_row_q4_K_cuda;
        case 13:
            return dequantize_row_q5_K_cuda;
        case 14:
            return dequantize_row_q6_K_cuda;
        case 16:
            return dequantize_row_iq2_xxs_cuda;
        case 17:
            return dequantize_row_iq2_xs_cuda;
        default:
            return nullptr;
    }
}

// GEMV
template <int qk, int qr, dequantize_kernel_t dequantize_kernel>
static __global__ void dequantize_mul_mat_vec(const void * __restrict__ vx, const dfloat * __restrict__ y, dfloat * __restrict__ dst, const int ncols, const int nrows) {
    // qk = quantized weights per x block
    // qr = number of quantized weights per data value in x block
    const int row = blockIdx.x*blockDim.y + threadIdx.y;

    if (row >= nrows) {
        return;
    }

    const int tid = threadIdx.x;

    const int iter_stride = 2*GGML_CUDA_DMMV_X;
    const int vals_per_iter = iter_stride / WARP_SIZE; // num quantized vals per thread and i iter
    const int y_offset = qr == 1 ? 1 : qk/2;

    half2 tmp = __floats2half2_rn(0.0f, 0.0f); // two sums for f16 to take advantage of half2 intrinsics

    for (int i = 0; i < ncols; i += iter_stride) {
        const int col = i + vals_per_iter*tid;
        const int ib = (row*ncols + col)/qk; // x block index
        const int iqs = (col%qk)/qr; // x quant index
        const int iybs = col - col%qk; // y block start index

// processing >2 values per i iter is faster for fast GPUs
#pragma unroll
        for (int j = 0; j < vals_per_iter; j += 2) {
            // process 2 vals per j iter

            // dequantize
            // for qr = 2 the iqs needs to increase by 1 per j iter because 2 weights per data val
            dfloat2 v;
            dequantize_kernel(vx, ib, iqs + j/qr, v);

            // matrix multiplication
            // for qr = 2 the y index needs to increase by 1 per j iter because of y_offset = qk/2
            tmp = __hadd2(tmp, __hmul2(v, {
                y[iybs + iqs + j/qr + 0],
                y[iybs + iqs + j/qr + y_offset]
            }));
        }
    }

    // sum up partial sums and write back result
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp = __hadd2(tmp, __shfl_xor_sync(0xffffffff, tmp, mask, 32));
    }

    if (tid == 0) {
        dst[row] = __hadd(tmp.x, tmp.y);
    }
}


static __global__ void dequantize_mul_mat_vec_q2_k(const void * __restrict__ vx, const dfloat * __restrict__ yy, dfloat * __restrict__ dst, const int ncols, int nrows) {

    static_assert(16%K_QUANTS_PER_ITERATION == 0, "16 must be divisible by K_QUANTS_PER_ITERATION");

    const int row = blockIdx.x*blockDim.y + threadIdx.y;
    if (row > nrows) return;

    const int num_blocks_per_row = ncols / QK_K;
    const int ib0 = row*num_blocks_per_row;

    const block_q2_K * x = (const block_q2_K *)vx + ib0;

    float tmp = 0; // partial sum for thread in warp

    const int tid = threadIdx.x/K_QUANTS_PER_ITERATION;  // 0...31 or 0...15
    const int ix  = threadIdx.x%K_QUANTS_PER_ITERATION;  // 0 or 0,1

    const int step = 16/K_QUANTS_PER_ITERATION;

    const int im = tid/step;                             // 0 or 1. 0 computes 0..., 1 computes 128...
    const int in = tid - step*im;                        // 0...15 or 0...7

    const int l0 = K_QUANTS_PER_ITERATION*in;            // 0...15 or 0...14 in steps of 2
    const int q_offset = 32*im + l0;
    const int s_offset = 8*im;
    const int y_offset = 128*im + l0;

    uint32_t aux[4];
    const uint8_t * d = (const uint8_t *)aux;
    const uint8_t * m = (const uint8_t *)(aux + 2);

    for (int i = ix; i < num_blocks_per_row; i += K_QUANTS_PER_ITERATION) {

        const half    * y = yy + i * QK_K + y_offset;
        const uint8_t * q = x[i].qs + q_offset;

        const float dall = __low2float(x[i].dm);
        const float dmin = __high2float(x[i].dm);

        const uint32_t * a = (const uint32_t *)(x[i].scales + s_offset);
        aux[0] = a[0] & 0x0f0f0f0f;
        aux[1] = a[1] & 0x0f0f0f0f;
        aux[2] = (a[0] >> 4) & 0x0f0f0f0f;
        aux[3] = (a[1] >> 4) & 0x0f0f0f0f;

        float sum1 = 0, sum2 = 0;
        for (int l = 0; l < K_QUANTS_PER_ITERATION; ++l) {
            sum1 += __half2float(y[l+ 0]) * d[0] * ((q[l+ 0] >> 0) & 3)
                  + __half2float(y[l+32]) * d[2] * ((q[l+ 0] >> 2) & 3)
                  + __half2float(y[l+64]) * d[4] * ((q[l+ 0] >> 4) & 3)
                  + __half2float(y[l+96]) * d[6] * ((q[l+ 0] >> 6) & 3)
                  + __half2float(y[l+16]) * d[1] * ((q[l+16] >> 0) & 3)
                  + __half2float(y[l+48]) * d[3] * ((q[l+16] >> 2) & 3)
                  + __half2float(y[l+80]) * d[5] * ((q[l+16] >> 4) & 3)
                  +__half2float(y[l+112]) * d[7] * ((q[l+16] >> 6) & 3);
            sum2 += __half2float(y[l+ 0]) * m[0] + __half2float(y[l+32]) * m[2] + __half2float(y[l+64]) * m[4] + __half2float(y[ l+96]) * m[6]
                  + __half2float(y[l+16]) * m[1] + __half2float(y[l+48]) * m[3] + __half2float(y[l+80]) * m[5] + __half2float(y[l+112]) * m[7];

        }
        tmp += dall * sum1 - dmin * sum2;

    }

    // sum up partial sums and write back result
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp += __shfl_xor_sync(0xffffffff, tmp, mask, 32);
    }

    if (threadIdx.x == 0) {
        dst[row] = __float2half(tmp);
    }
}

static __global__ void dequantize_mul_mat_vec_q3_k(const void * __restrict__ vx, const dfloat * __restrict__ yy, dfloat * __restrict__ dst, const int ncols, int nrows) {

    const int row = blockIdx.x*blockDim.y + threadIdx.y;
    if (row > nrows) return;

    const int num_blocks_per_row = ncols / QK_K;
    const int ib0 = row*num_blocks_per_row;

    const block_q3_K * x = (const block_q3_K *)vx + ib0;

    float tmp = 0; // partial sum for thread in warp

    const uint16_t kmask1 = 0x0303;
    const uint16_t kmask2 = 0x0f0f;

    const int tid = threadIdx.x/K_QUANTS_PER_ITERATION;  // 0...31 or 0...16
    const int ix  = threadIdx.x%K_QUANTS_PER_ITERATION;  // 0 or 0,1

    const int n  = K_QUANTS_PER_ITERATION;               // iterations in the inner loop
    const int step = 16/K_QUANTS_PER_ITERATION;
    const int im = tid/step;                             // 0 or 1. 0 computes 0..., 1 computes 128...
    const int in = tid - step*im;                        // 0....15 or 0...7

    const uint8_t m = 1 << (4*im);

    const int l0 = n*in;                                 // 0...15 or 0...14 in steps of 2
    const int q_offset =  32*im + l0;
    const int y_offset = 128*im + l0;

    uint16_t utmp[4];
    const int8_t * s = (const int8_t *)utmp;

    const uint16_t s_shift = 4*im;

    for (int i = ix; i < num_blocks_per_row; i += K_QUANTS_PER_ITERATION) {

        const half    * y  = yy + i * QK_K + y_offset;
        const uint8_t * q = x[i].qs + q_offset;
        const uint8_t * h = x[i].hmask + l0;

        const uint16_t * a = (const uint16_t *)x[i].scales;
        utmp[0] = ((a[0] >> s_shift) & kmask2) | (((a[4] >> (s_shift + 0)) & kmask1) << 4);
        utmp[1] = ((a[1] >> s_shift) & kmask2) | (((a[5] >> (s_shift + 0)) & kmask1) << 4);
        utmp[2] = ((a[2] >> s_shift) & kmask2) | (((a[4] >> (s_shift + 2)) & kmask1) << 4);
        utmp[3] = ((a[3] >> s_shift) & kmask2) | (((a[5] >> (s_shift + 2)) & kmask1) << 4);

        const float d = __half2float(x[i].d);

        float sum = 0;
        for (int l = 0; l < n; ++l) {
            sum += __half2float(y[l+ 0]) * (s[0] - 32) * (((q[l] >> 0) & 3) - (h[l] & (m << 0) ? 0 : 4))
                 + __half2float(y[l+32]) * (s[2] - 32) * (((q[l] >> 2) & 3) - (h[l] & (m << 1) ? 0 : 4))
                 + __half2float(y[l+64]) * (s[4] - 32) * (((q[l] >> 4) & 3) - (h[l] & (m << 2) ? 0 : 4))
                 + __half2float(y[l+96]) * (s[6] - 32) * (((q[l] >> 6) & 3) - (h[l] & (m << 3) ? 0 : 4));
            sum += __half2float(y[l+16]) * (s[1] - 32) * (((q[l+16] >> 0) & 3) - (h[l+16] & (m << 0) ? 0 : 4))
                 + __half2float(y[l+48]) * (s[3] - 32) * (((q[l+16] >> 2) & 3) - (h[l+16] & (m << 1) ? 0 : 4))
                 + __half2float(y[l+80]) * (s[5] - 32) * (((q[l+16] >> 4) & 3) - (h[l+16] & (m << 2) ? 0 : 4))
                + __half2float(y[l+112]) * (s[7] - 32) * (((q[l+16] >> 6) & 3) - (h[l+16] & (m << 3) ? 0 : 4));
        }
        tmp += d * sum;

    }

    // sum up partial sums and write back result
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp += __shfl_xor_sync(0xffffffff, tmp, mask, 32);
    }

    if (threadIdx.x == 0) {
        dst[row] = __float2half(tmp);
    }
}

static __global__ void dequantize_mul_mat_vec_q4_k(const void * __restrict__ vx, const dfloat * __restrict__ yy, dfloat * __restrict__ dst, const int ncols, int nrows) {

    const int row = blockIdx.x*blockDim.y + threadIdx.y;
    if (row > nrows) return;
    const int num_blocks_per_row = ncols / QK_K;
    const int ib0 = row*num_blocks_per_row;

    const block_q4_K * x = (const block_q4_K *)vx + ib0;

    const uint16_t kmask1 = 0x3f3f;
    const uint16_t kmask2 = 0x0f0f;
    const uint16_t kmask3 = 0xc0c0;

    const int tid = threadIdx.x/K_QUANTS_PER_ITERATION;  // 0...31 or 0...16
    const int ix  = threadIdx.x%K_QUANTS_PER_ITERATION;  // 0 or 0,1

    const int step = 8/K_QUANTS_PER_ITERATION;           // 8 or 4

    const int il  = tid/step;                            // 0...3
    const int ir  = tid - step*il;                       // 0...7 or 0...3
    const int n   = 2 * K_QUANTS_PER_ITERATION;          // 2 or 4

    const int im = il/2;  // 0 or 1. 0 computes 0,32 + 128,160, 1 computes 64,96 + 192,224
    const int in = il%2;

    const int l0 = n*(2*ir + in);
    const int q_offset = 32*im + l0;
    const int y_offset = 64*im + l0;

    uint16_t aux[4];
    const uint8_t * sc = (const uint8_t *)aux;

#if K_QUANTS_PER_ITERATION == 2
    uint32_t q32[4];
    const uint8_t * q4 = (const uint8_t *)q32;
#else
    uint16_t q16[4];
    const uint8_t * q4 = (const uint8_t *)q16;
#endif

    float tmp = 0; // partial sum for thread in warp

    for (int i = ix; i < num_blocks_per_row; i += K_QUANTS_PER_ITERATION) {

        const half   * y1 = yy + i*QK_K + y_offset;
        const half   * y2 = y1 + 128;

        const float dall = __low2float(x[i].dm);
        const float dmin = __high2float(x[i].dm);

        const uint16_t * a = (const uint16_t *)x[i].scales;
        aux[0] = a[im+0] & kmask1;
        aux[1] = a[im+2] & kmask1;
        aux[2] = ((a[im+4] >> 0) & kmask2) | ((a[im+0] & kmask3) >> 2);
        aux[3] = ((a[im+4] >> 4) & kmask2) | ((a[im+2] & kmask3) >> 2);

#if K_QUANTS_PER_ITERATION == 2
        const uint32_t * q1 = (const uint32_t *)(x[i].qs + q_offset);
        const uint32_t * q2 = q1 + 16;

        q32[0] = q1[0] & 0x0f0f0f0f;
        q32[1] = q1[0] & 0xf0f0f0f0;
        q32[2] = q2[0] & 0x0f0f0f0f;
        q32[3] = q2[0] & 0xf0f0f0f0;

        float4 s = {0.f, 0.f, 0.f, 0.f};
        float smin = 0;
        for (int l = 0; l < 4; ++l) {
            s.x += __half2float(y1[l]) * q4[l+0]; s.y += __half2float(y1[l+32]) * q4[l+ 4];
            s.z += __half2float(y2[l]) * q4[l+8]; s.w += __half2float(y2[l+32]) * q4[l+12];
            smin += __half2float(y1[l]) * sc[2] + __half2float(y1[l+32]) * sc[3] + __half2float(y2[l]) * sc[6] + __half2float(y2[l+32]) * sc[7];
        }
        tmp += dall * (s.x * sc[0] + s.y * sc[1] * 1.f/16.f + s.z * sc[4] + s.w * sc[5] * 1.f/16.f) - dmin * smin;
#else
        const uint16_t * q1 = (const uint16_t *)(x[i].qs + q_offset);
        const uint16_t * q2 = q1 + 32;

        q16[0] = q1[0] & 0x0f0f;
        q16[1] = q1[0] & 0xf0f0;
        q16[2] = q2[0] & 0x0f0f;
        q16[3] = q2[0] & 0xf0f0;

        float4 s = {0.f, 0.f, 0.f, 0.f};
        float smin = 0;
        for (int l = 0; l < 2; ++l) {
            s.x += __half2float(y1[l]) * q4[l+0]; s.y += __half2float(y1[l+32]) * q4[l+2];
            s.z += __half2float(y2[l]) * q4[l+4]; s.w += __half2float(y2[l+32]) * q4[l+6];
            smin += __half2float(y1[l]) * sc[2] + __half2float(y1[l+32]) * sc[3] + __half2float(y2[l]) * sc[6] + __half2float(y2[l+32]) * sc[7];
        }
        tmp += dall * (s.x * sc[0] + s.y * sc[1] * 1.f/16.f + s.z * sc[4] + s.w * sc[5] * 1.f/16.f) - dmin * smin;
#endif

    }

    // sum up partial sums and write back result
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp += __shfl_xor_sync(0xffffffff, tmp, mask, 32);
    }

    if (tid == 0) {
        dst[row] = __float2half(tmp);
    }
}

static __global__ void dequantize_mul_mat_vec_q5_k(const void * __restrict__ vx, const dfloat * __restrict__ yy, dfloat * __restrict__ dst, const int ncols) {

    const int row = blockIdx.x;
    const int num_blocks_per_row = ncols / QK_K;
    const int ib0 = row*num_blocks_per_row;

    const block_q5_K * x = (const block_q5_K *)vx + ib0;

    float tmp = 0; // partial sum for thread in warp

    const uint16_t kmask1 = 0x3f3f;
    const uint16_t kmask2 = 0x0f0f;
    const uint16_t kmask3 = 0xc0c0;

    const int tid = threadIdx.x/2;  // 0...15
    const int ix  = threadIdx.x%2;

    const int il  = tid/4;     // 0...3
    const int ir  = tid - 4*il;// 0...3
    const int n   = 2;

    const int im = il/2;  // 0 or 1. 0 computes 0,32 + 128,160, 1 computes 64,96 + 192,224
    const int in = il%2;

    const int l0 = n*(2*ir + in);
    const int q_offset = 32*im + l0;
    const int y_offset = 64*im + l0;

    const uint8_t hm1  = 1 << (2*im);
    const uint8_t hm2  = hm1 << 4;

    uint16_t aux[4];
    const uint8_t * sc = (const uint8_t *)aux;

    uint16_t q16[8];
    const uint8_t * q4 = (const uint8_t *)q16;

    for (int i = ix; i < num_blocks_per_row; i += 2) {

        const uint8_t * ql1 = x[i].qs + q_offset;
        const uint8_t * qh  = x[i].qh + l0;
        const half    * y1  = yy + i*QK_K + y_offset;
        const half    * y2  = y1 + 128;

        const float dall = __low2float(x[i].dm);
        const float dmin = __high2float(x[i].dm);

        const uint16_t * a = (const uint16_t *)x[i].scales;
        aux[0] = a[im+0] & kmask1;
        aux[1] = a[im+2] & kmask1;
        aux[2] = ((a[im+4] >> 0) & kmask2) | ((a[im+0] & kmask3) >> 2);
        aux[3] = ((a[im+4] >> 4) & kmask2) | ((a[im+2] & kmask3) >> 2);

        float4 sum = {0.f, 0.f, 0.f, 0.f};
        float smin = 0;
        const uint16_t * q1 = (const uint16_t *)ql1;
        const uint16_t * q2 = q1 + 32;
        q16[0] = q1[0] & 0x0f0f;
        q16[1] = q1[8] & 0x0f0f;
        q16[2] = (q1[0] >> 4) & 0x0f0f;
        q16[3] = (q1[8] >> 4) & 0x0f0f;
        q16[4] = q2[0] & 0x0f0f;
        q16[5] = q2[8] & 0x0f0f;
        q16[6] = (q2[0] >> 4) & 0x0f0f;
        q16[7] = (q2[8] >> 4) & 0x0f0f;
        for (int l = 0; l < n; ++l) {
            sum.x += __half2float(y1[l+ 0]) * (q4[l +0] + (qh[l+ 0] & (hm1 << 0) ? 16 : 0))
                   + __half2float(y1[l+16]) * (q4[l +2] + (qh[l+16] & (hm1 << 0) ? 16 : 0));
            sum.y += __half2float(y1[l+32]) * (q4[l +4] + (qh[l+ 0] & (hm1 << 1) ? 16 : 0))
                   + __half2float(y1[l+48]) * (q4[l +6] + (qh[l+16] & (hm1 << 1) ? 16 : 0));
            sum.z += __half2float(y2[l+ 0]) * (q4[l +8] + (qh[l+ 0] & (hm2 << 0) ? 16 : 0))
                   + __half2float(y2[l+16]) * (q4[l+10] + (qh[l+16] & (hm2 << 0) ? 16 : 0));
            sum.w += __half2float(y2[l+32]) * (q4[l+12] + (qh[l+ 0] & (hm2 << 1) ? 16 : 0))
                   + __half2float(y2[l+48]) * (q4[l+14] + (qh[l+16] & (hm2 << 1) ? 16 : 0));
            smin += (__half2float(y1[l]) + __half2float(y1[l+16])) * sc[2] + (__half2float(y1[l+32]) + __half2float(y1[l+48])) * sc[3]
                  + (__half2float(y2[l]) + __half2float(y2[l+16])) * sc[6] + (__half2float(y2[l+32]) + __half2float(y2[l+48])) * sc[7];
        }
        tmp += dall * (sum.x * sc[0] + sum.y * sc[1] + sum.z * sc[4] + sum.w * sc[5]) - dmin * smin;
    }

    // sum up partial sums and write back result
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp += __shfl_xor_sync(0xffffffff, tmp, mask, 32);
    }

    if (threadIdx.x == 0) {
        dst[row] = __float2half(tmp);
    }
}

static __global__ void dequantize_mul_mat_vec_q6_k(const void * __restrict__ vx, const dfloat * __restrict__ yy, dfloat * __restrict__ dst, const int ncols, int nrows) {

    static_assert(16%K_QUANTS_PER_ITERATION == 0, "16 must be divisible by K_QUANTS_PER_ITERATION");

    const int row = blockIdx.x*blockDim.y + threadIdx.y;
    if (row > nrows) return;

    const int num_blocks_per_row = ncols / QK_K;
    const int ib0 = row*num_blocks_per_row;

    const block_q6_K * x = (const block_q6_K *)vx + ib0;

    const int tid = threadIdx.x/K_QUANTS_PER_ITERATION;  // 0...31 or 0...16
    const int ix  = threadIdx.x%K_QUANTS_PER_ITERATION;  // 0 or 0, 1

    const int step = 16/K_QUANTS_PER_ITERATION;          // 16 or 8

    const int im = tid/step;                             // 0 or 1. 0 computes 0..., 1 computes 128...
    const int in = tid - step*im;                        // 0...15 or 0...7

#if K_QUANTS_PER_ITERATION == 1
    const int l0 = K_QUANTS_PER_ITERATION*in;            // 0...15
    const int is = 0;
#else
    const int l0 = 4 * in;                               // 0, 4, 8, ..., 28
    const int is = in / 4;
#endif
    const int ql_offset = 64*im + l0;
    const int qh_offset = 32*im + l0;
    const int s_offset  =  8*im + is;
    const int y_offset = 128*im + l0;

    float tmp = 0; // partial sum for thread in warp

    for (int i = ix; i < num_blocks_per_row; i += K_QUANTS_PER_ITERATION) {

        const half    * y  = yy + i * QK_K + y_offset;
        const uint8_t * ql = x[i].ql + ql_offset;
        const uint8_t * qh = x[i].qh + qh_offset;
        const int8_t  * s  = x[i].scales + s_offset;

        const float d = __half2float(x[i].d);

#if K_QUANTS_PER_ITERATION == 1
        float sum = __half2float(y[ 0]) * s[0] * d * ((int8_t)((ql[ 0] & 0xF) | ((qh[ 0] & 0x03) << 4)) - 32)
                  + __half2float(y[16]) * s[1] * d * ((int8_t)((ql[16] & 0xF) | ((qh[16] & 0x03) << 4)) - 32)
                  + __half2float(y[32]) * s[2] * d * ((int8_t)((ql[32] & 0xF) | ((qh[ 0] & 0x0c) << 2)) - 32)
                  + __half2float(y[48]) * s[3] * d * ((int8_t)((ql[48] & 0xF) | ((qh[16] & 0x0c) << 2)) - 32)
                  + __half2float(y[64]) * s[4] * d * ((int8_t)((ql[ 0]  >> 4) | ((qh[ 0] & 0x30) >> 0)) - 32)
                  + __half2float(y[80]) * s[5] * d * ((int8_t)((ql[16]  >> 4) | ((qh[16] & 0x30) >> 0)) - 32)
                  + __half2float(y[96]) * s[6] * d * ((int8_t)((ql[32]  >> 4) | ((qh[ 0] & 0xc0) >> 2)) - 32)
                  +__half2float(y[112]) * s[7] * d * ((int8_t)((ql[48]  >> 4) | ((qh[16] & 0xc0) >> 2)) - 32);
        tmp += sum;
#else
        float sum = 0;
        for (int l = 0; l < 4; ++l) {
            sum += __half2float(y[l+ 0]) * s[0] * d * ((int8_t)((ql[l+ 0] & 0xF) | (((qh[l] >> 0) & 3) << 4)) - 32)
                 + __half2float(y[l+32]) * s[2] * d * ((int8_t)((ql[l+32] & 0xF) | (((qh[l] >> 2) & 3) << 4)) - 32)
                 + __half2float(y[l+64]) * s[4] * d * ((int8_t)((ql[l+ 0]  >> 4) | (((qh[l] >> 4) & 3) << 4)) - 32)
                 + __half2float(y[l+96]) * s[6] * d * ((int8_t)((ql[l+32]  >> 4) | (((qh[l] >> 6) & 3) << 4)) - 32);
        }
        tmp += sum;
#endif

    }

    // sum up partial sums and write back result
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp += __shfl_xor_sync(0xffffffff, tmp, mask, 32);
    }

    if (tid == 0) {
        dst[row] = __float2half(tmp);
    }
}

static __global__ void dequantize_mul_mat_vec_iq2_xxs(const void * __restrict__ vx, const dfloat * __restrict__ yy, dfloat * __restrict__ dst, const int ncols, int nrows) {
    const int row = blockIdx.x*blockDim.y + threadIdx.y;
    if (row > nrows) return;

    const int num_blocks_per_row = ncols / QK_K;
    const int ib0 = row*num_blocks_per_row;

    const block_iq2_xxs * x = (const block_iq2_xxs *)vx + ib0;

    float tmp = 0; // partial sum for thread in warp

    const int tid = threadIdx.x/4;
    const int ix  = threadIdx.x%4;

    const int q_offset = tid * 4;
    const int y_offset = tid * 32;

    for (int i = ix; i < num_blocks_per_row; i += 4) {

        const half    * y = yy + i * QK_K + y_offset;
        const uint16_t * q = x[i].qs + q_offset;

        const uint8_t  * aux8 = (const uint8_t *)q;
        uint32_t aux32 = q[2] | (q[3] << 16);
        float sumi = 0;
        for (int l = 0; l < 4; ++l) {
            const uint8_t * grid = (const uint8_t *)(iq2xxs_grid + aux8[l]);
            const uint8_t  signs = ksigns_iq2xs[aux32 & 127];
            for (int j = 0; j < 8; ++j) {
                sumi += __half2float(y[j]) * grid[j] * (signs & kmask_iq2xs[j] ? -1 : 1);
            }
            y += 8;
            aux32 >>= 7;
        }
        tmp += sumi * __half2float(x[i].d) * (0.5f + aux32) * 0.25f;;
    }

    // sum up partial sums and write back result
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp += __shfl_xor_sync(0xffffffff, tmp, mask, 32);
    }

    if (threadIdx.x == 0) {
        dst[row] = __float2half(tmp);
    }
}

static __global__ void dequantize_mul_mat_vec_iq2_xs(const void * __restrict__ vx, const dfloat * __restrict__ yy, dfloat * __restrict__ dst, const int ncols, int nrows) {
    const int row = blockIdx.x*blockDim.y + threadIdx.y;
    if (row > nrows) return;

    const int num_blocks_per_row = ncols / QK_K;
    const int ib0 = row*num_blocks_per_row;

    const block_iq2_xs * x = (const block_iq2_xs *)vx + ib0;

    float tmp = 0; // partial sum for thread in warp

    const int tid = threadIdx.x/4;
    const int ix  = threadIdx.x%4;

    const int q_offset = tid * 4;
    const int s_offset = tid;
    const int y_offset = tid * 32;

    for (int i = ix; i < num_blocks_per_row; i += 4) {
        const half    * y = yy + i * QK_K + y_offset;
        const uint16_t * q = x[i].qs + q_offset;
        const uint8_t ls1 = x[i].scales[s_offset] & 0xf;
        const uint8_t ls2 = x[i].scales[s_offset] >>  4;

        float sumi1 = 0;
        for (int l = 0; l < 2; ++l) {
            const uint8_t * grid = (const uint8_t *)(iq2xs_grid + (q[l] & 511));
            const uint8_t  signs = ksigns_iq2xs[q[l] >> 9];
            for (int j = 0; j < 8; ++j) {
                sumi1 += __half2float(y[j]) * grid[j] * (signs & kmask_iq2xs[j] ? -1 : 1);
            }
            y += 8;
        }
        float sumi2 = 0;
        for (int l = 2; l < 4; ++l) {
            const uint8_t * grid = (const uint8_t *)(iq2xs_grid + (q[l] & 511));
            const uint8_t  signs = ksigns_iq2xs[q[l] >> 9];
            for (int j = 0; j < 8; ++j) {
                sumi2 += __half2float(y[j]) * grid[j] * (signs & kmask_iq2xs[j] ? -1 : 1);
            }
            y += 8;
        }
        const float d = __half2float(x[i].d) * 0.25f;
        tmp += d * ((0.5f + ls1) * sumi1 + (0.5f + ls2) * sumi2);;
    }

    // sum up partial sums and write back result
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp += __shfl_xor_sync(0xffffffff, tmp, mask, 32);
    }

    if (threadIdx.x == 0) {
        dst[row] = __float2half(tmp);
    }
}

static void dequantize_mul_mat_vec_q4_0_cuda(const void * vx, const dfloat * y, dfloat * dst, const int ncols, const int nrows, hipStream_t stream) {
    const int block_num_y = (nrows + GGML_CUDA_MMV_Y - 1) / GGML_CUDA_MMV_Y;
    // the number of rows may exceed maximum grid size in the y or z dimensions, use the x dimension instead
    const dim3 block_nums(block_num_y, 1, 1);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_MMV_Y, 1);
    dequantize_mul_mat_vec<QK4_0, QR4_0, dequantize_q4_0>
        <<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

static void dequantize_mul_mat_vec_q4_1_cuda(const void * vx, const dfloat * y, dfloat * dst, const int ncols, const int nrows, hipStream_t stream) {
    const int block_num_y = (nrows + GGML_CUDA_MMV_Y - 1) / GGML_CUDA_MMV_Y;
    const dim3 block_nums(block_num_y, 1, 1);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_MMV_Y, 1);
    dequantize_mul_mat_vec<QK4_1, QR4_1, dequantize_q4_1>
        <<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

static void dequantize_mul_mat_vec_q5_0_cuda(const void * vx, const dfloat * y, dfloat * dst, const int ncols, const int nrows, hipStream_t stream) {
    const int block_num_y = (nrows + GGML_CUDA_MMV_Y - 1) / GGML_CUDA_MMV_Y;
    const dim3 block_nums(block_num_y, 1, 1);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_MMV_Y, 1);
    dequantize_mul_mat_vec<QK5_0, QR5_0, dequantize_q5_0>
        <<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

static void dequantize_mul_mat_vec_q5_1_cuda(const void * vx, const dfloat * y, dfloat * dst, const int ncols, const int nrows, hipStream_t stream) {
    const int block_num_y = (nrows + GGML_CUDA_MMV_Y - 1) / GGML_CUDA_MMV_Y;
    const dim3 block_nums(block_num_y, 1, 1);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_MMV_Y, 1);
    dequantize_mul_mat_vec<QK5_1, QR5_1, dequantize_q5_1>
        <<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

static void dequantize_mul_mat_vec_q8_0_cuda(const void * vx, const dfloat * y, dfloat * dst, const int ncols, const int nrows, hipStream_t stream) {
    const int block_num_y = (nrows + GGML_CUDA_MMV_Y - 1) / GGML_CUDA_MMV_Y;
    const dim3 block_nums(block_num_y, 1, 1);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_MMV_Y, 1);
    dequantize_mul_mat_vec<QK8_0, QR8_0, dequantize_q8_0>
        <<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

static void dequantize_mul_mat_vec_q2_K_cuda(const void * vx, const dfloat * y, dfloat * dst, const int ncols, const int nrows, hipStream_t stream) {
    const int ny = 2; // very slightly faster than 1 even when K_QUANTS_PER_ITERATION = 2
    const int block_num_y = (nrows + ny - 1) / ny;
    const dim3 block_nums(block_num_y, 1, 1);
    const dim3 block_dims(32, ny, 1);
    dequantize_mul_mat_vec_q2_k<<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

static void dequantize_mul_mat_vec_q3_K_cuda(const void * vx, const dfloat * y, dfloat * dst, const int ncols, const int nrows, hipStream_t stream) {
    const int ny = 2 / K_QUANTS_PER_ITERATION;
    const int block_num_y = (nrows + ny - 1) / ny;
    const dim3 block_nums(block_num_y, 1, 1);
    const dim3 block_dims(32, ny, 1);
    dequantize_mul_mat_vec_q3_k<<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

static void dequantize_mul_mat_vec_q4_K_cuda(const void * vx, const dfloat * y, dfloat * dst, const int ncols, const int nrows, hipStream_t stream) {
    const int ny = 2 / K_QUANTS_PER_ITERATION;
    const int block_num_y = (nrows + ny - 1) / ny;
    const dim3 block_nums(block_num_y, 1, 1);
    const dim3 block_dims(32, ny, 1);
    dequantize_mul_mat_vec_q4_k<<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

static void dequantize_mul_mat_vec_q5_K_cuda(const void * vx, const dfloat * y, dfloat * dst, const int ncols, const int nrows, hipStream_t stream) {
    const dim3 block_dims(32, 1, 1);
    dequantize_mul_mat_vec_q5_k<<<nrows, block_dims, 0, stream>>>(vx, y, dst, ncols);
}

static void dequantize_mul_mat_vec_q6_K_cuda(const void * vx, const dfloat * y, dfloat * dst, const int ncols, const int nrows, hipStream_t stream) {
    const int ny = 2 / K_QUANTS_PER_ITERATION;
    const int block_num_y = (nrows + ny - 1) / ny;
    const dim3 block_nums(block_num_y, 1, 1);
    const dim3 block_dims(32, ny, 1);
    dequantize_mul_mat_vec_q6_k<<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

static void dequantize_mul_mat_vec_iq2_xxs_cuda(const void * vx, const dfloat * y, dfloat * dst, const int ncols, const int nrows, hipStream_t stream) {
    const dim3 block_dims(32, 1, 1);
    dequantize_mul_mat_vec_iq2_xxs<<<nrows, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

static void dequantize_mul_mat_vec_iq2_xs_cuda(const void * vx, const dfloat * y, dfloat * dst, const int ncols, const int nrows, hipStream_t stream) {
    const dim3 block_dims(32, 1, 1);
    dequantize_mul_mat_vec_iq2_xs<<<nrows, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

torch::Tensor ggml_dequantize(
    torch::Tensor W,   // quant weight
    int8_t type,
    int64_t m,
    int64_t n
){
    const at::cuda::OptionalCUDAGuard device_guard(device_of(W));
    auto options = torch::TensorOptions().dtype(torch::kFloat16).device(W.device());
    at::Tensor DW = torch::empty({m, n}, options);
    hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
    const to_fp16_cuda_t to_fp16_cuda = ggml_get_to_fp16_cuda(type);
    to_fp16_cuda(
        (void*)W.data_ptr(), (half*)DW.data_ptr(), m * n, stream
    );
    return DW;
}

torch::Tensor ggml_mul_mat_vec(
    torch::Tensor W,  // quant weight
    torch::Tensor X,  // input
    int8_t type,
    int64_t row
){
    size_t col = X.sizes()[1];
    // printf("%d %d\n", col, row);
    const at::cuda::OptionalCUDAGuard device_guard(device_of(X));
    auto options = torch::TensorOptions().dtype(torch::kFloat16).device(W.device());
    at::Tensor Y = torch::empty({1, row}, options);
    hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
    switch (type) {
        case 2:
            dequantize_mul_mat_vec_q4_0_cuda((void*)W.data_ptr(), (half*)X.data_ptr(), (half*)Y.data_ptr(), col, row, stream);
            break;
        case 3:
            dequantize_mul_mat_vec_q4_1_cuda((void*)W.data_ptr(), (half*)X.data_ptr(), (half*)Y.data_ptr(), col, row, stream);
            break;
        case 6:
            dequantize_mul_mat_vec_q5_0_cuda((void*)W.data_ptr(), (half*)X.data_ptr(), (half*)Y.data_ptr(), col, row, stream);
            break;
        case 7:
            dequantize_mul_mat_vec_q5_1_cuda((void*)W.data_ptr(), (half*)X.data_ptr(), (half*)Y.data_ptr(), col, row, stream);
            break;
        case 8:
            dequantize_mul_mat_vec_q8_0_cuda((void*)W.data_ptr(), (half*)X.data_ptr(), (half*)Y.data_ptr(), col, row, stream);
            break;
        case 10:
            dequantize_mul_mat_vec_q2_K_cuda((void*)W.data_ptr(), (half*)X.data_ptr(), (half*)Y.data_ptr(), col, row, stream);
            break;
        case 11:
            dequantize_mul_mat_vec_q3_K_cuda((void*)W.data_ptr(), (half*)X.data_ptr(), (half*)Y.data_ptr(), col, row, stream);
            break;
        case 12:
            dequantize_mul_mat_vec_q4_K_cuda((void*)W.data_ptr(), (half*)X.data_ptr(), (half*)Y.data_ptr(), col, row, stream);
            break;
        case 13:
            dequantize_mul_mat_vec_q5_K_cuda((void*)W.data_ptr(), (half*)X.data_ptr(), (half*)Y.data_ptr(), col, row, stream);
            break;
        case 14:
            dequantize_mul_mat_vec_q6_K_cuda((void*)W.data_ptr(), (half*)X.data_ptr(), (half*)Y.data_ptr(), col, row, stream);
            break;
        case 16:
            dequantize_mul_mat_vec_iq2_xxs_cuda((void*)W.data_ptr(), (half*)X.data_ptr(), (half*)Y.data_ptr(), col, row, stream);
            break;
        case 17:
            dequantize_mul_mat_vec_iq2_xs_cuda((void*)W.data_ptr(), (half*)X.data_ptr(), (half*)Y.data_ptr(), col, row, stream);
            break;
    }
    return Y;
}
