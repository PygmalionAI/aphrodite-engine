#include "hip/hip_runtime.h"
/*
 * Adapted from https://github.com/turboderp/exllamav2
 * Copyright (c) 2024 turboderp
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */
#include <torch/extension.h>
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>

#include "q_matrix.cuh"
#include "matrix_view.cuh"

#include "quant/qdq_2.cuh"
#include "quant/qdq_3.cuh"
#include "quant/qdq_4.cuh"
#include "quant/qdq_5.cuh"
#include "quant/qdq_6.cuh"
#include "quant/qdq_8.cuh"


namespace aphrodite {
namespace exl2 {

#define BLOCK_KN_SIZE 128

#define THREADS_X 32
#define THREADS_Y 32

#define DIVIDE(x, size) (((x) + (size) - 1) / (size))

// Shuffle quantized data on load

__global__ void shuffle_kernel
(
    uint32_t* __restrict__ b_q_weight,
    const int size_k,
    const int size_n,
    const int rows_8,
    const int rows_6,
    const int rows_5,
    const int rows_4,
    const int rows_3,
    const int rows_2
)
{
    int n = blockIdx.x * THREADS_X + threadIdx.x;
    if (n >= size_n) return;
    int k = 0;
    uint32_t* b_ptr = b_q_weight + n;
    while (k < rows_8) { shuffle_8bit_4 (b_ptr, size_n); b_ptr += 1 * size_n; k +=  4; }
    while (k < rows_6) { shuffle_6bit_16(b_ptr, size_n); b_ptr += 3 * size_n; k += 16; }
    while (k < rows_5) { shuffle_5bit_32(b_ptr, size_n); b_ptr += 5 * size_n; k += 32; }
    while (k < rows_4) { shuffle_4bit_8 (b_ptr, size_n); b_ptr += 1 * size_n; k +=  8; }
    while (k < rows_3) { shuffle_3bit_32(b_ptr, size_n); b_ptr += 3 * size_n; k += 32; }
    while (k < rows_2) { shuffle_2bit_16(b_ptr, size_n); b_ptr += 1 * size_n; k += 16; }
}


// QMatrix constructor

QMatrix::QMatrix
(
    const int _device,
    const int _height,
    const int _width,
    const int _groups,

    uint32_t* _q_weight,
    uint16_t* _q_perm,
    uint16_t* _q_invperm,
    uint32_t* _q_scale,
    half* _q_scale_max,
    uint16_t* _q_groups,
    uint16_t* _q_group_map
):
    device(_device),
    height(_height),
    width(_width),
    groups(_groups)
{
    hipSetDevice(device);

    failed = false;

    cuda_q_weight = _q_weight;
    cuda_q_perm = _q_perm;
    cuda_q_invperm = _q_invperm;
    cuda_q_scale = _q_scale;
    cuda_q_scale_max = _q_scale_max;
    cuda_q_groups = _q_groups;
    cuda_q_group_map = _q_group_map;

    // Create group map

    rows_8 = 0;
    rows_6 = 0;
    rows_5 = 0;
    rows_4 = 0;
    rows_3 = 0;
    rows_2 = 0;

    {
        uint16_t* cpu_q_groups = (uint16_t*)calloc(groups * 2, sizeof(uint16_t));
        hipMemcpy(cpu_q_groups, cuda_q_groups, groups * 2 * sizeof(uint16_t), hipMemcpyDeviceToHost);

        int row = 0;
        for (int i = 0; i < groups; i++)
        {
            int bits = cpu_q_groups[i * 2];

            int rows;
            if (i < groups - 1)
            {
                int qrows = cpu_q_groups[i * 2 + 3] - cpu_q_groups[i * 2 + 1];
                rows = qrows * 32 / bits;
            }
            else rows = height - row;

            if (bits == 8) rows_8 += rows;
            if (bits == 6) rows_6 += rows;
            if (bits == 5) rows_5 += rows;
            if (bits == 4) rows_4 += rows;
            if (bits == 3) rows_3 += rows;
            if (bits == 2) rows_2 += rows;
            row += rows;
        }

        free(cpu_q_groups);

        rows_6 += rows_8;
        rows_5 += rows_6;
        rows_4 += rows_5;
        rows_3 += rows_4;
        rows_2 += rows_3;
    }

    // Shuffle quantized data

    dim3 blockDim, gridDim;
    blockDim.x = THREADS_X;
    blockDim.y = 1;
    gridDim.x = DIVIDE(width, THREADS_X);
    gridDim.y = 1;
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();

    shuffle_kernel<<<gridDim, blockDim, 0, stream>>>(
        cuda_q_weight, height, width, rows_8, rows_6, rows_5, rows_4, rows_3, rows_2);
}

QMatrix::~QMatrix()
{
}


// Reconstruct b[k,n]

__global__ void reconstruct_kernel
(
    const uint32_t* __restrict__ b_q_weight,
    const uint16_t* __restrict__ b_q_perm,
    const uint32_t* __restrict__ b_q_scale,
    const half* __restrict__ b_q_scale_max,
    const uint16_t* __restrict__ b_q_group_map,
    const int size_k,
    const int size_n,
    //const int groupsize,
    const int groups,
    half* __restrict__ b,
    const int rows_8,
    const int rows_6,
    const int rows_5,
    const int rows_4,
    const int rows_3,
    const int rows_2
)
{
    MatrixView_half_rw b_(b, size_k, size_n);
    MatrixView_q4_row b_q_scale_(b_q_scale, groups, size_n);

    int offset_k = BLOCK_KN_SIZE * blockIdx.y;
    int offset_n = BLOCK_KN_SIZE * blockIdx.x;

    // Preload remapping table

    int t = threadIdx.x;
    __shared__ uint16_t perm[BLOCK_KN_SIZE];
    if (offset_k + t < size_k)
        perm[t] = b_q_perm[offset_k + t];

    // Column

    int n = offset_n + t;
    if (n >= size_n) return;

    // Find initial group

    // int group = offset_k / groupsize;
    int group = b_q_group_map[offset_k * 2];

    int pre_rows_8 = min(rows_8, offset_k);
    int pre_rows_6 = offset_k > rows_8 ? min(rows_6, offset_k) - rows_8 : 0;
    int pre_rows_5 = offset_k > rows_6 ? min(rows_5, offset_k) - rows_6 : 0;
    int pre_rows_4 = offset_k > rows_5 ? min(rows_4, offset_k) - rows_5 : 0;
    int pre_rows_3 = offset_k > rows_4 ? min(rows_3, offset_k) - rows_4 : 0;
    int pre_rows_2 = offset_k > rows_3 ? min(rows_2, offset_k) - rows_3 : 0;
    int qk = 0;
    qk += pre_rows_8 / 32 * 8;
    qk += pre_rows_6 / 32 * 6;
    qk += pre_rows_5 / 32 * 5;
    qk += pre_rows_4 / 32 * 4;
    qk += pre_rows_3 / 32 * 3;
    qk += pre_rows_2 / 32 * 2;

    const uint32_t* b_ptr = b_q_weight + qk * size_n + n;

    half qs_h = dq_scale(b_q_scale_.item(group, n), b_q_scale_max[group]);
    half2 qs_h2 = __halves2half2(qs_h, qs_h);
    int nextgroup = offset_k + b_q_group_map[offset_k * 2 + 1];

    int end_k = min(offset_k + BLOCK_KN_SIZE, size_k);
    int k = offset_k;
    int lk = 0;

    __syncthreads();

    while (k < rows_8 && k < end_k)
    {
        if (k == nextgroup) { group++; qs_h = dq_scale(b_q_scale_.item(group, n), b_q_scale_max[group]); nextgroup += b_q_group_map[k * 2 + 1]; qs_h2 = __halves2half2(qs_h, qs_h); }
        for (int p = 0; p < 4; p++)
        {
            half2 dq[4];
            uint32_t q_0 = *b_ptr; b_ptr += size_n;
            uint32_t q_1 = *b_ptr; b_ptr += size_n;
            dequant_8bit_8(q_0, q_1, dq, size_n);
            for (int j = 0; j < 4; j++) dq[j] = __hmul2(dq[j], qs_h2);
            half* dqh = (half*) dq;
            for (int j = 0; j < 8; j++) b_.set(perm[lk++], n, dqh[j]);
        }
        k += 32;
    }

    while (k < rows_6 && k < end_k)
    {
        if (k == nextgroup) { group++; qs_h = dq_scale(b_q_scale_.item(group, n), b_q_scale_max[group]); nextgroup += b_q_group_map[k * 2 + 1]; qs_h2 = __halves2half2(qs_h, qs_h); }
        for (int p = 0; p < 2; p++)
        {
            half2 dq[8];
            uint32_t q_0 = *b_ptr; b_ptr += size_n;
            uint32_t q_1 = *b_ptr; b_ptr += size_n;
            uint32_t q_2 = *b_ptr; b_ptr += size_n;
            dequant_6bit_16(q_0, q_1, q_2, dq, size_n);
            for (int j = 0; j < 8; j++) dq[j] = __hmul2(dq[j], qs_h2);
            half* dqh = (half*) dq;
            for (int j = 0; j < 16; j++) b_.set(perm[lk++], n, dqh[j]);
        }
        k += 32;
    }

    while (k < rows_5 && k < end_k)
    {
        if (k == nextgroup) { group++; qs_h = dq_scale(b_q_scale_.item(group, n), b_q_scale_max[group]); nextgroup += b_q_group_map[k * 2 + 1]; qs_h2 = __halves2half2(qs_h, qs_h); }
        for (int p = 0; p < 1; p++)
        {
            half2 dq[16];
            uint32_t q_0 = *b_ptr; b_ptr += size_n;
            uint32_t q_1 = *b_ptr; b_ptr += size_n;
            uint32_t q_2 = *b_ptr; b_ptr += size_n;
            uint32_t q_3 = *b_ptr; b_ptr += size_n;
            uint32_t q_4 = *b_ptr; b_ptr += size_n;
            dequant_5bit_32(q_0, q_1, q_2, q_3, q_4, dq, size_n);
            for (int j = 0; j < 16; j++) dq[j] = __hmul2(dq[j], qs_h2);
            half* dqh = (half*) dq;
            for (int j = 0; j < 32; j++) b_.set(perm[lk++], n, dqh[j]);
        }
        k += 32;
    }

    while (k < rows_4 && k < end_k)
    {
        if (k == nextgroup) { group++; qs_h = dq_scale(b_q_scale_.item(group, n), b_q_scale_max[group]); nextgroup += b_q_group_map[k * 2 + 1]; qs_h2 = __halves2half2(qs_h, qs_h); }
        for (int p = 0; p < 4; p++)
        {
            half2 dq[4];
            uint32_t q_0 = *b_ptr; b_ptr += size_n;
            dequant_4bit_8(q_0, dq, size_n);
            for (int j = 0; j < 4; j++) dq[j] = __hmul2(dq[j], qs_h2);
            half* dqh = (half*) dq;
            for (int j = 0; j < 8; j++) b_.set(perm[lk++], n, dqh[j]);
        }
        k += 32;
    }

    while (k < rows_3 && k < end_k)
    {
        if (k == nextgroup) { group++; qs_h = dq_scale(b_q_scale_.item(group, n), b_q_scale_max[group]); nextgroup += b_q_group_map[k * 2 + 1]; qs_h2 = __halves2half2(qs_h, qs_h); }
        for (int p = 0; p < 1; p++)
        {
            half2 dq[16];
            uint32_t q_0 = *b_ptr; b_ptr += size_n;
            uint32_t q_1 = *b_ptr; b_ptr += size_n;
            uint32_t q_2 = *b_ptr; b_ptr += size_n;
            dequant_3bit_32(q_0, q_1, q_2, dq, size_n);
            for (int j = 0; j < 16; j++) dq[j] = __hmul2(dq[j], qs_h2);
            half* dqh = (half*) dq;
            for (int j = 0; j < 32; j++) b_.set(perm[lk++], n, dqh[j]);
        }
        k += 32;
    }

    while (k < rows_2 && k < end_k)
    {
        if (k == nextgroup) { group++; qs_h = dq_scale(b_q_scale_.item(group, n), b_q_scale_max[group]); nextgroup += b_q_group_map[k * 2 + 1]; qs_h2 = __halves2half2(qs_h, qs_h); }
        for (int p = 0; p < 1; p++)
        {
            half2 dq[8];
            uint32_t q_0 = *b_ptr; b_ptr += size_n;
            dequant_2bit_16(q_0, dq, size_n);
            for (int j = 0; j < 8; j++) dq[j] = __hmul2(dq[j], qs_h2);
            half* dqh = (half*) dq;
            for (int j = 0; j < 16; j++) b_.set(perm[lk++], n, dqh[j]);
        }
        k += 16;
    }
}

void QMatrix::reconstruct(half* out)
{
    dim3 blockDim, gridDim;
    blockDim.x = BLOCK_KN_SIZE;
    blockDim.y = 1;
    gridDim.y = DIVIDE(height, BLOCK_KN_SIZE);

    {
        gridDim.x = DIVIDE(width, BLOCK_KN_SIZE);
        reconstruct_kernel<<<gridDim, blockDim>>>
        (
            cuda_q_weight,
            cuda_q_perm,
            cuda_q_scale,
            cuda_q_scale_max,
            cuda_q_group_map,
            height,
            width,
            //groupsize,
            groups,
            out,
            rows_8,
            rows_6,
            rows_5,
            rows_4,
            rows_3,
            rows_2
        );
    }
}

}  // namespace exl2
}  // namespace aphrodite