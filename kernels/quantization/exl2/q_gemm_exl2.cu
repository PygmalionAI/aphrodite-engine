#include "hip/hip_runtime.h"
/*
 * Adapted from https://github.com/turboderp/exllamav2
 * Copyright (c) 2024 turboderp
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */
#include <torch/extension.h>
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>

#include "q_matrix.cuh"
#include "matrix_view.cuh"
#include "quant/qdq_2.cuh"
#include "quant/qdq_3.cuh"
#include "quant/qdq_4.cuh"
#include "quant/qdq_5.cuh"
#include "quant/qdq_6.cuh"
#include "quant/qdq_8.cuh"
#include "q_gemm_kernel.cuh"

namespace aphrodite {
namespace exl2 {

#define MAX_Q_GEMM_ROWS 32
#define EXL2_BLOCK_KN_SIZE 64
#define EXL2_BLOCK_M_SIZE_MAX 8
#define EXL2_MAX_GROUPS_IN_BLOCK (EXL2_BLOCK_KN_SIZE / 32)

#define DIVIDE(x, size) (((x) + (size) - 1) / (size))

void gemm_half_q_half_cuda_part
(
    const half* a,
    QMatrix* b,
    half* c,
    int size_m,
    int size_n,
    int size_k,
    int m_count,
    bool clear
)
{
    {
        dim3 blockDim, gridDim;
        blockDim.x = EXL2_BLOCK_KN_SIZE;
        blockDim.y = 1;
        blockDim.z = 1;
        gridDim.x = DIVIDE(size_n, EXL2_BLOCK_KN_SIZE * 4);
        gridDim.y = DIVIDE(size_m, m_count);
        gridDim.z = DIVIDE(size_k, EXL2_BLOCK_KN_SIZE);

        fp_gemm_half_q_half_kernel kernel = pick_gemm_half_q_half_kernel(m_count);
        const hipStream_t stream = at::cuda::getCurrentCUDAStream();

        kernel<<<gridDim, blockDim, 0, stream>>>
        (
            a,
            b->cuda_q_weight,
            b->cuda_q_scale,
            b->cuda_q_scale_max,
            c,
            size_m,
            size_n,
            size_k,
            b->groups,
            b->cuda_q_group_map,
            b->cuda_q_perm,
            b->rows_8,
            b->rows_6,
            b->rows_5,
            b->rows_4,
            b->rows_3,
            b->rows_2,
            clear
        );
    }

}

void gemm_half_q_half_cuda
(
    hipblasHandle_t cublas_handle,
    const half* a,
    QMatrix* b,
    half* c,
    int size_m,
    int size_n,
    int size_k,
    bool clear,
    half* temp_dq
)
{
    if (size_m > MAX_Q_GEMM_ROWS)
    {
        // Reconstruct FP16 matrix, then cuBLAS
        b->reconstruct(temp_dq);

        //hipblasSetMathMode(cublas_handle, HIPBLAS_TENSOR_OP_MATH);

        const half alpha = __float2half(1.0f);
        const half beta = clear ? __float2half(0.0f) : __float2half(1.0f);
        hipblasHgemm(cublas_handle,
                    HIPBLAS_OP_N,
                    HIPBLAS_OP_N,
                    size_n, size_m, size_k,
                    &alpha, temp_dq, size_n,
                            a,       size_k,
                    &beta,  c,       size_n);
    }
    else
    {
        // Quantized matmul

        int block_m_size_max = EXL2_BLOCK_M_SIZE_MAX;
        int max_chunks = size_m / block_m_size_max;
        int last_chunk = max_chunks * block_m_size_max;
        int last_chunk_size = size_m - last_chunk;

        if (max_chunks)
        {
            gemm_half_q_half_cuda_part(a, b, c, last_chunk, size_n, size_k, block_m_size_max, clear);
        }

        if (last_chunk_size)
        {
            gemm_half_q_half_cuda_part(a + last_chunk * size_k, b, c + last_chunk * size_n, last_chunk_size, size_n, size_k, last_chunk_size, clear);
        }
    }
}

}  // namespace exl2
}  // namespace aphrodite

torch::Tensor exl2_gemm
(
    torch::Tensor a,
    uintptr_t b
)
{
    const at::cuda::OptionalCUDAGuard device_guard(device_of(a));
    aphrodite::exl2::QMatrix* qm = reinterpret_cast<aphrodite::exl2::QMatrix*> (b);

    auto options = torch::TensorOptions().dtype(a.dtype()).device(a.device());
    at::Tensor c = torch::empty({a.size(0), qm->width}, options);
    at::Tensor temp_dq = torch::empty({a.size(1), qm->width}, options);

    aphrodite::exl2::gemm_half_q_half_cuda
    (
        at::cuda::getCurrentCUDABlasHandle(),
        (const half*) a.data_ptr(),
        qm,
        (half*) c.data_ptr(),
        c.size(0),  // m
        c.size(1),  // n
        a.size(1),  // k
        true,
        (half*) temp_dq.data_ptr()
    );
    return c;
}

uintptr_t make_q_matrix
(
    torch::Tensor q_weight,
    torch::Tensor q_perm,
    torch::Tensor q_invperm,
    torch::Tensor q_scale,
    torch::Tensor q_scale_max,
    torch::Tensor q_groups,
    torch::Tensor q_group_map
)
{
    const at::cuda::OptionalCUDAGuard device_guard(device_of(q_weight));
    int device = q_weight.device().index();
    int width = q_weight.size(1);
    int groups = q_scale.size(0);
    int height = q_invperm.size(0);

    aphrodite::exl2::QMatrix* m = new aphrodite::exl2::QMatrix
    (
        device,
        height,
        width,
        groups,
        (uint32_t*) q_weight.data_ptr(),
        (uint16_t*) q_perm.data_ptr(),
        (uint16_t*) q_invperm.data_ptr(),
        (uint32_t*) q_scale.data_ptr(),
        (half*) q_scale_max.data_ptr(),
        (uint16_t*) q_groups.data_ptr(),
        (uint16_t*) q_group_map.data_ptr()
    );
    return reinterpret_cast<uintptr_t>(m);
}