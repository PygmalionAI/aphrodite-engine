/******************************************************************************
 * Copyright (c) 2023, Tri Dao.
 ******************************************************************************/

// Split into multiple files to compile in parallel

#include "selective_scan_fwd_kernel.cuh"

template void selective_scan_fwd_cuda<at::BFloat16, float>(
    SSMParamsBase& params, hipStream_t stream);
template void selective_scan_fwd_cuda<at::BFloat16, complex_t>(
    SSMParamsBase& params, hipStream_t stream);