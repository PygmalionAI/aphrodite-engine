/******************************************************************************
 * Copyright (c) 2023, Tri Dao.
 ******************************************************************************/

// Split into multiple files to compile in parallel

#include "selective_scan_fwd_kernel.cuh"

template void selective_scan_fwd_cuda<at::Half, float>(SSMParamsBase& params,
                                                       hipStream_t stream);
template void selective_scan_fwd_cuda<at::Half, complex_t>(
    SSMParamsBase& params, hipStream_t stream);